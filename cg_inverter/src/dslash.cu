#include <cstdio>
#include <cmath>
#include <assert.h>
#include <chrono>
#include <mpi.h>
#include "qcu.h"
#include <hip/hip_runtime.h>
#include "qcu_complex.cuh"
#include "qcu_dslash.cuh"
#include "qcu_macro.cuh"
#include "qcu_complex_computation.cuh"
#include "qcu_point.cuh"
#include "qcu_communicator.cuh"
#include "qcu_clover_dslash.cuh"
#include "qcu_wilson_dslash_neo.cuh"
#include "qcu_wilson_dslash.cuh"
#include "qcu_shift_storage_complex.cuh"
#include "qcu_wilson_dslash_new_new.cuh"
#include <iostream>
using std::cout;
using std::endl;
#define qcuPrint() { \
    printf("function %s line %d...\n", __FUNCTION__, __LINE__); \
}


void* qcu_gauge;
void loadQcuGauge(void* gauge, QcuParam *param) {
  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];

  checkCudaErrors(hipMalloc(&qcu_gauge, sizeof(double) * Nd * Lx * Ly * Lz * Lt * (Nc-1) * Nc * 2));
  shiftGaugeStorageTwoDouble(qcu_gauge, gauge, TO_COALESCE, Lx, Ly, Lz, Lt);
}






void dslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity) {
  // getDeviceInfo();
  // parity ---- invert_flag

  // cloverDslashOneRound(fermion_out, fermion_in, gauge, param, 0);
  // cloverDslashOneRound(fermion_out, fermion_in, gauge, param, parity);
  // fullCloverDslashOneRound(fermion_out, fermion_in, gauge, param, 0);
  // wilsonDslashOneRound(fermion_out, fermion_in, gauge, param, parity);
  // callWilsonDslash(fermion_out, fermion_in, gauge, param, parity, 0);

  // callWilsonDslash(fermion_out, fermion_in, qcu_gauge, param, parity, 0);
  // callWilsonDslashFull(fermion_out, fermion_in, gauge, param, parity, 0);


  // callWilsonDslashNaive(fermion_out, fermion_in, gauge, param, parity, 0);
  // callNop(fermion_out, fermion_in, gauge, param, parity, 0);
  // calculateNaiveOnlyMemoryAccessing(fermion_out, fermion_in, gauge, param, parity, 0);
  // callNewDslash(fermion_out, fermion_in, gauge, param, parity, 0);
  callNewDslashCoalesced(fermion_out, fermion_in, gauge, param, parity, 0);
}
void fullDslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int dagger_flag) {
  fullCloverDslashOneRound (fermion_out, fermion_in, gauge, param, dagger_flag);
}
