#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <assert.h>
#include <chrono>
#include <mpi.h>
#include "qcu.h"
#include <hip/hip_runtime.h>
#include "qcu_complex.cuh"
#include "qcu_complex_computation.cuh"
#include "qcu_macro.cuh"
#include "qcu_clover_dslash.cuh"
#include "qcu_communicator.cuh"
#include "qcu_wilson_dslash_neo.cuh"

// #define DEBUG


extern MPICommunicator *mpi_comm;

/**
 * @brief clear the Complex vector named vec of vector_length elements to zero (kernel function)
 * 
 * @param vec 
 * @param vector_length 
 * @return void
 */
static __global__ void clearVectorKernel(void* vec, int vector_length) {
  int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
  int vol = blockDim.x * gridDim.x;
  Complex* src = static_cast<Complex*>(vec);

  if (thread_id >= vector_length) {
    return;
  }

  for (int i = thread_id; i < vector_length; i += vol) {
    src[i].clear2Zero();
  }
}

/**
 * @brief clear the Complex vector named vec of vector_length elements to zero (host function)
 * 
 * @param vec 
 * @param vector_length 
 * @return void
 */
static void clear_vector (void* vec, int vector_length) {
  int block_size = MAX_BLOCK_SIZE;
  int grid_size = (vector_length + block_size * Ns * Nc - 1) / (block_size * Ns * Nc);
  clearVectorKernel<<<grid_size, block_size>>>(vec, vector_length);
  qcuCudaDeviceSynchronize();
}


/**
 * @brief use this function to do one time MmV, when what to calc odd x
 * 
 * @param output_Ax result which means matrix A multiply vector x
 * @param input_x input vector x
 * @param temp_vec1 temporary space
 * @param temp_vec2 temporary space
 * @param gauge gauge
 * @param d_kappa kappa device pointer
 * @param param size of Lx Ly Lz Lt
 * @param dagger_flag 0 means no dagger, 1 means dagger
 * @param kappa kappa (double)
 */
void odd_matrix_mul_vector (void* output_Ax, void* input_x, void* temp_vec1, void* temp_vec2, void* gauge, void* d_kappa, QcuParam *param, int dagger_flag = 0, double kappa = 1.0) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  int parity;     // when use Doe, parity is odd which means 1, else 0
  Complex h_coeff;

  qcuCudaMemcpy(output_Ax, input_x, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);

  // part1 begin
  parity = 1;
  cloverVectorHalf (output_Ax, nullptr, gauge, param, parity);
  // part1 end

  // part2 begin
  parity = 0;
  callWilsonDslashNaive(temp_vec1, input_x, gauge, param, parity, dagger_flag);
  parity = 0;
  invertCloverDslashHalf (temp_vec1, nullptr, gauge, param, parity);  // clover invert
  parity = 1;
  callWilsonDslashNaive(temp_vec2, temp_vec1, gauge, param, parity, dagger_flag);
  // part2 end

  h_coeff = Complex(-kappa * kappa, 0);
  qcuCudaMemcpy(d_kappa, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  // saxpy
  mpi_comm->interprocess_saxpy_barrier(temp_vec2, output_Ax, d_kappa, half_vol);  // coeff temp2 + x --->x
}

void full_odd_matrix_mul_vector (void* output_Ax, void* input_x, void* temp_vec1, void* temp_vec2, void* temp_vec3, void* gauge, void* d_kappa, QcuParam *param, double kappa = 1.0) {

  int dagger_flag;
  dagger_flag = 0;
  odd_matrix_mul_vector (temp_vec3, input_x, temp_vec1, temp_vec2, gauge, \
                          d_kappa, param, dagger_flag, kappa);
  dagger_flag = 1;
  odd_matrix_mul_vector (output_Ax, temp_vec3, temp_vec1, temp_vec2, gauge, \
                          d_kappa, param, dagger_flag, kappa);
}
// current_b is temporary
bool if_even_converge(void* current_x, void* current_b_buffer, void* target_b, \
                    void* temp_vec1, void* temp_vec2, void* temp_vec3, \
                    void* gauge, void* d_kappa, void* d_coeff, \
                    void* d_norm1, void* d_norm2, QcuParam *param, double kappa = 1.0
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex h_coeff;
  double h_norm1; // norm(target_b)
  double h_norm2; // norm(target_b - current_b)

  int parity = 0;

  cloverVectorHalf (current_b_buffer, nullptr, gauge, param, parity);  // Ax ---> temp_vec1

  gpu_vector_norm2 (target_b, temp_vec3, half_vol, d_norm1);

  qcuCudaMemcpy (temp_vec2, target_b, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);     // target_b -----> temp_vec2
  h_coeff = Complex(-1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(current_b_buffer, temp_vec2, d_coeff, \
                half_vol); // temp_vec2 <--- target_b - current_b

  gpu_vector_norm2(temp_vec2, temp_vec3, half_vol, d_norm2);
  qcuCudaMemcpy(&h_norm1, d_norm1, sizeof(double), hipMemcpyDeviceToHost);
  qcuCudaMemcpy(&h_norm2, d_norm2, sizeof(double), hipMemcpyDeviceToHost);
  return (h_norm2 / h_norm1 < 1e-13); // which means converge
}

// current_b is temporary
bool if_odd_converge(void* current_x, void* current_b_buffer, void* target_b, \
                    void* temp_vec1, void* temp_vec2, void* temp_vec3, \
                    void* gauge, void* d_kappa, void* d_coeff, \
                    void* d_norm1, void* d_norm2, QcuParam *param, double kappa = 1.0
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex h_coeff;
  double h_norm1; // norm(target_b)
  double h_norm2; // norm(target_b - current_b)

  full_odd_matrix_mul_vector (current_b_buffer, current_x, \
                temp_vec1, temp_vec2, temp_vec3, gauge, d_kappa, param, kappa);

  gpu_vector_norm2 (target_b, temp_vec3, half_vol, d_norm1);

  qcuCudaMemcpy (temp_vec2, target_b, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);     // target_b -----> temp_vec2
  h_coeff = Complex(-1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(current_b_buffer, temp_vec2, d_coeff, \
                half_vol); // temp_vec2 <--- target_b - current_b

  gpu_vector_norm2(temp_vec2, temp_vec3, half_vol, d_norm2);
  qcuCudaMemcpy(&h_norm1, d_norm1, sizeof(double), hipMemcpyDeviceToHost);
  qcuCudaMemcpy(&h_norm2, d_norm2, sizeof(double), hipMemcpyDeviceToHost);
#ifdef DEBUG
  printf("difference %.32lf, norm1 = %lf, norm2 = %lf\n", \
              h_norm2 / h_norm1, h_norm1, h_norm2);
#endif
  return (h_norm2 / h_norm1 < 1e-13); // which means converge
}

bool odd_cg_iter(void* iter_x_odd, void* target_b, void* resid_vec, void* p_vec, \
        void* temp_vec1, void* temp_vec2, void* temp_vec3, void* temp_vec4, void* temp_vec5, \
        void* gauge, QcuParam *param, double kappa, void* d_kappa, \
        void* d_alpha, void* d_beta, void* d_denominator, void* d_numerator, \
        void* d_coeff, void* d_norm1, void* d_norm2
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex alpha;
  Complex beta;
  Complex denominator;
  Complex numerator;
  Complex one(1,0);

  // <r, r>--->denominator
  mpi_comm->interprocess_inner_prod_barrier(resid_vec, resid_vec, \
                                    d_numerator, half_vol);  // <r, r> --> d_numerator


  full_odd_matrix_mul_vector (temp_vec4, p_vec, temp_vec1, \
                    temp_vec2, temp_vec3, gauge, d_kappa, param, kappa);
                    // Ap --->temp_vec4


  mpi_comm->interprocess_inner_prod_barrier(p_vec, temp_vec4, d_denominator, \
                    half_vol);  // <p, Ap> --> d_denominator

  qcuCudaMemcpy(&numerator, d_numerator, sizeof(Complex), hipMemcpyDeviceToHost);
  qcuCudaMemcpy(&denominator, d_denominator, sizeof(Complex), hipMemcpyDeviceToHost);
// #ifdef DEBUG
//   printf(RED"");
//   printf("numerator %lf %lf\n", numerator.real(), numerator.imag());
//   printf("denominator %lf %lf\n", denominator.real(), denominator.imag());
//   printf(CLR"");
// #endif

  alpha = numerator / denominator;
  qcuCudaMemcpy(d_alpha, &alpha, sizeof(Complex), hipMemcpyHostToDevice);

  mpi_comm->interprocess_saxpy_barrier(p_vec, iter_x_odd, d_alpha, half_vol); // x = x + \alpha p

  qcuCudaMemcpy(temp_vec1, resid_vec, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice); // copy r to temp_vec1  r'=r

  alpha = alpha * Complex(-1, 0);
  qcuCudaMemcpy(d_alpha, &alpha, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec4, temp_vec1, d_alpha, half_vol); // temp_vec4 = Ap, r'=r'-\alpha Ap------>temp_vec1

  if (if_odd_converge(iter_x_odd, temp_vec5, \
                      target_b, temp_vec2, temp_vec3,\
                      temp_vec4, gauge, d_kappa, d_coeff, \
                      d_norm1, d_norm2, param, kappa)
  ) { // donnot use temp_vec1 !!!!
    return true;
  }

  // <r, r> is in numerator
  mpi_comm->interprocess_inner_prod_barrier(temp_vec1, temp_vec1, \
                                            d_denominator, half_vol);  // <r', r'>

  qcuCudaMemcpy(&denominator, d_denominator, sizeof(Complex), hipMemcpyDeviceToHost);
  beta = denominator / numerator;
  qcuCudaMemcpy(d_beta, &beta, sizeof(Complex), hipMemcpyHostToDevice);
  // p = r' + \beta p
  gpu_sclar_multiply_vector (p_vec, d_beta, half_vol); // p_vec = \beta p_vec
  one = Complex(1, 0);
  qcuCudaMemcpy(d_coeff, &one, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec1, p_vec, d_coeff, half_vol); // p <-- r' + \beta p

  qcuCudaMemcpy(resid_vec, temp_vec1, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);  // r <--- r'

  return false;
}




bool even_cg_iter(void* iter_x_odd, void* target_b, void* resid_vec, void* p_vec, \
        void* temp_vec1, void* temp_vec2, void* temp_vec3, void* temp_vec4, void* temp_vec5, \
        void* gauge, QcuParam *param, double kappa, void* d_kappa, \
        void* d_alpha, void* d_beta, void* d_denominator, void* d_numerator, \
        void* d_coeff, void* d_norm1, void* d_norm2
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex alpha;
  Complex beta;
  Complex denominator;
  Complex numerator;
  Complex one(1,0);

  // <r, r>--->denominator
  mpi_comm->interprocess_inner_prod_barrier(resid_vec, resid_vec, \
                                    d_numerator, half_vol);  // <r, r> --> d_numerator


  qcuCudaMemcpy(temp_vec4, p_vec, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);
  cloverVectorHalf (temp_vec4, nullptr, gauge, param, 0);  // Ap --->temp_vec4


  mpi_comm->interprocess_inner_prod_barrier(p_vec, temp_vec4, d_denominator, \
                    half_vol);  // <p, Ap> --> d_denominator

  qcuCudaMemcpy(&numerator, d_numerator, sizeof(Complex), hipMemcpyDeviceToHost);
  qcuCudaMemcpy(&denominator, d_denominator, sizeof(Complex), hipMemcpyDeviceToHost);

  alpha = numerator / denominator;
  qcuCudaMemcpy(d_alpha, &alpha, sizeof(Complex), hipMemcpyHostToDevice);

  mpi_comm->interprocess_saxpy_barrier(p_vec, iter_x_odd, d_alpha, half_vol); // x = x + \alpha p

  qcuCudaMemcpy(temp_vec1, resid_vec, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice); // copy r to temp_vec1  r'=r

  alpha = alpha * Complex(-1, 0);
  qcuCudaMemcpy(d_alpha, &alpha, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec4, temp_vec1, d_alpha, half_vol); // temp_vec4 = Ap, r'=r-\alpha Ap------>temp_vec1

  if (if_even_converge(iter_x_odd, temp_vec5, \
                      target_b, temp_vec2, temp_vec3,\
                      temp_vec4, gauge, d_kappa, d_coeff, \
                      d_norm1, d_norm2, param, kappa)
  ) { // donnot use temp_vec1 !!!!
    return true;
  }

  // <r, r> is in numerator
  mpi_comm->interprocess_inner_prod_barrier(temp_vec1, temp_vec1, \
                                            d_denominator, half_vol);  // <r', r'>

  qcuCudaMemcpy(&denominator, d_denominator, sizeof(Complex), hipMemcpyDeviceToHost);
  beta = denominator / numerator;
  qcuCudaMemcpy(d_beta, &beta, sizeof(Complex), hipMemcpyHostToDevice);
  // p = r' + \beta p
  gpu_sclar_multiply_vector (p_vec, d_beta, half_vol); // p_vec = \beta p_vec
  one = Complex(1, 0);
  qcuCudaMemcpy(d_coeff, &one, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec1, p_vec, d_coeff, half_vol); // p <-- r' + \beta p

  qcuCudaMemcpy(resid_vec, temp_vec1, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);  // r <--- r'

  return false;
}



// cg_even
bool even_cg_inverter (void* iter_x_even, void* target_b, void* resid_vec, void* p_vec,
  void* temp_vec1, void* temp_vec2, void* temp_vec3, void* temp_vec4, void* temp_vec5,\
  void* gauge, QcuParam *param, double kappa, void* d_kappa, \
  void* d_alpha, void* d_beta, void* d_denominator, void* d_numerator, void* d_coeff, void* d_norm1, void* d_norm2
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  int parity;
  bool if_converge = false;
  Complex h_coeff;

  clear_vector (iter_x_even, half_vol * Ns * Nc);  // x <-- 0
  // b - Ax --->r
  qcuCudaMemcpy (resid_vec, target_b, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);      // r <-- b

  qcuCudaMemcpy (temp_vec1, iter_x_even, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);  // x-->temp_vec1
  parity = 0;
  cloverVectorHalf (temp_vec1, nullptr, gauge, param, parity);  // Ax ---> temp_vec1

  if_converge =  if_even_converge(iter_x_even, temp_vec5, target_b, temp_vec1, \
                                  temp_vec2, temp_vec3, gauge, d_kappa, d_coeff, \
                                  d_norm1, d_norm2, param, kappa);
  if (if_converge) {
    return if_converge;
  }
  // then   r--->p
  qcuCudaMemcpy(p_vec, resid_vec, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);


  for (int i = 0; i < half_vol; i++) {
    if_converge = even_cg_iter(iter_x_even, target_b, resid_vec, p_vec, \
                          temp_vec1, temp_vec2, temp_vec3, temp_vec4, temp_vec5, \
                          gauge, param, kappa, d_kappa, d_alpha, d_beta, \
                          d_denominator, d_numerator, d_coeff, d_norm1, d_norm2);
    if (if_converge) {
      printf("even cg success! %d iterations\n", i+1);
      break;
    }
  }

  return if_converge;
}


// cg_odd
bool odd_cg_inverter (void* iter_x_odd, void* target_b, void* resid_vec, void* p_vec, \
  void* temp_vec1, void* temp_vec2, void* temp_vec3, void* temp_vec4, void* temp_vec5,\
  void* gauge, QcuParam *param, double kappa, void* d_kappa, \
  void* d_alpha, void* d_beta, void* d_denominator, void* d_numerator, void* d_coeff, void* d_norm1, void* d_norm2
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  bool converge = false;
  Complex h_coeff;

  clear_vector (iter_x_odd, half_vol * Ns * Nc);  // x <-- 0
  // b - Ax --->r
  qcuCudaMemcpy (resid_vec, target_b, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);      // r <-- b
  // second: Ax ---> temp_vec4
  full_odd_matrix_mul_vector (temp_vec4, iter_x_odd, temp_vec1, \
                            temp_vec2, temp_vec3, gauge, d_kappa, param, kappa);

  h_coeff = Complex(-1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec4, resid_vec, d_coeff, \
                                        half_vol);  // last: r <-- b-Ax


  // If converge return x
  if (if_odd_converge(iter_x_odd, temp_vec4, target_b, temp_vec1, temp_vec2, temp_vec3, gauge, d_kappa, d_coeff, d_norm1, d_norm2, param, kappa)) {
    printf("cg suceess!\n");
    goto odd_cg_end;
  }

#ifdef DEBUG
  printf(RED"first iteration passed\n");
  printf(CLR"");
#endif
  // p <-- r
  qcuCudaMemcpy(p_vec, resid_vec, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice);


  for (int i = 0; i < half_vol; i++) {
#ifdef DEBUG
    printf(RED"iteration %d\n", i+1);
    printf(CLR"");
#endif


  // test inner prod
// #ifdef DEBUG
//   Complex temp;
//   mpi_comm->interprocess_inner_prod_barrier(target_b, target_b, temp_vec5, half_vol);
//   qcuCudaMemcpy(&temp, temp_vec5, sizeof(Complex), hipMemcpyDeviceToHost);
//   printf(BLUE"temp.real = %lf, temp.imag = %lf\n", temp.real(), temp.imag());
//   printf(CLR"");
// #endif

    converge = odd_cg_iter(iter_x_odd, target_b, resid_vec, p_vec, \
                          temp_vec1, temp_vec2, temp_vec3, temp_vec4, temp_vec5, \
                          gauge, param, kappa, d_kappa, d_alpha, d_beta, \
                          d_denominator, d_numerator, d_coeff, d_norm1, d_norm2);

    if (converge) {
      printf("odd cg success! %d iterations\n", i+1);
      break;
    }
  }


odd_cg_end:
  return converge;
}

void generate_new_b_even (void* new_even_b, void* origin_even_b, void* res_odd_x, \
                          void* gauge, void* d_kappa, void* d_coeff, \
                        QcuParam *param, double kappa
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  int parity = 0;
  int dagger_flag = 0;
  Complex h_kappa(kappa, 0);
  Complex h_coeff;
  qcuCudaMemcpy(d_kappa, &h_kappa, sizeof(Complex), hipMemcpyHostToDevice);

  // D_{eo}x_{o} ----> new_even_b
  callWilsonDslashNaive (new_even_b, res_odd_x, gauge, param, parity, dagger_flag);
  // kappa D_{eo}x_{o} ----> new_even_b
  mpi_comm->interprocess_sax_barrier (new_even_b, d_kappa, half_vol);

  h_coeff = Complex(1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  // kappa D_{eo}x_{o} + even_b ----> new_even_b
  mpi_comm->interprocess_saxpy_barrier(origin_even_b, new_even_b, d_coeff, half_vol);
}


// modify b, half-length vector
void generate_new_b_odd(void* new_b, void* origin_odd_b, void* origin_even_b, \
                        void* temp_vec1, void* temp_vec2, void* temp_vec3, \
                        void* gauge, void* d_kappa, void* d_coeff, \
                        QcuParam *param, double kappa = 1.0
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex h_kappa;
  Complex h_coeff;
  int parity;
  int dagger_flag;

  // even b ----> temp_vec1
  qcuCudaMemcpy(temp_vec1, origin_even_b, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice);

  parity = 0;
  invertCloverDslashHalf (temp_vec1, nullptr, gauge, param, parity); // A^{-1}_{ee}b_{e} ---> temp_vec1

  parity = 1;
  dagger_flag = 0;
  callWilsonDslashNaive (new_b, temp_vec1, gauge, param, parity, dagger_flag); //  D_{oe}A^{-1}_{ee}b_{e} ----> new_b

  // kappa D_{oe}A^{-1}_{ee}b_{e}
  h_kappa = Complex(kappa, 0);
  qcuCudaMemcpy(d_kappa, &h_kappa, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_sax_barrier (new_b, d_kappa, half_vol);

  h_coeff = Complex(1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(origin_odd_b, new_b, d_coeff, half_vol);
}


void cg_inverter(void* b_vector, void* x_vector, void *gauge, QcuParam *param) {
  double kappa = 1.0;

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  bool if_end = false;
  // ptrs doesn't need new memory
  void* origin_even_b;
  void* origin_odd_b;
  void* even_x;
  void* odd_x;

  // ptrs need to allocate memory
  void* temp_vec1;
  void* temp_vec2;
  void* temp_vec3;
  void* temp_vec4;
  void* temp_vec5;

  void* p_vec;
  void* resid_vec;

  void* d_coeff;
  void* d_kappa;
  void* d_alpha;
  void* d_beta;
  void* d_denominator;
  void* d_numerator;
  void* d_norm1;
  void* d_norm2;
  void* new_b;

  int dagger_flag;

  origin_even_b = b_vector;
  origin_odd_b = static_cast<void*>(static_cast<Complex*>(b_vector) \
                  + half_vol * Ns * Nc);
  even_x = x_vector;
  odd_x = static_cast<void*>(static_cast<Complex*>(x_vector) + half_vol * Ns * Nc);

  // memory allocation
  qcuCudaMalloc(&temp_vec1, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&temp_vec2, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&temp_vec3, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&temp_vec4, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&temp_vec5, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&p_vec, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&resid_vec, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&d_coeff, sizeof(Complex));
  qcuCudaMalloc(&d_kappa, sizeof(Complex));

  qcuCudaMalloc(&d_alpha, sizeof(Complex));
  qcuCudaMalloc(&d_beta, sizeof(Complex));
  qcuCudaMalloc(&d_denominator, sizeof(Complex));
  qcuCudaMalloc(&d_numerator, sizeof(Complex));
  qcuCudaMalloc(&d_norm1, sizeof(Complex));
  qcuCudaMalloc(&d_norm2, sizeof(Complex));

  qcuCudaMalloc(&new_b, sizeof(Complex) * half_vol * Ns * Nc);



  // clear x, x <---0   void* odd_x 
  // clear_vector(odd_x, half_vol * Ns * Nc);
  // odd new_b
  generate_new_b_odd(temp_vec3, origin_odd_b, origin_even_b, temp_vec1, temp_vec2, temp_vec4, gauge, d_kappa, d_coeff, param, kappa);


// #ifdef DEBUG
//   double norm;
//   printf(BLUE"");
//   gpu_vector_norm2(temp_vec3, temp_vec5, half_vol, d_norm1);
//   qcuCudaMemcpy(&norm, d_norm1, sizeof(double), hipMemcpyDeviceToHost);
//   printf("norm2 of temp_vec3 is %lf\n", norm);
//   printf(CLR"");
// #endif


  // odd dagger D new_b
  dagger_flag = 1;
  odd_matrix_mul_vector (new_b, temp_vec3, temp_vec1, temp_vec2, \
                         gauge, d_kappa, param, dagger_flag, kappa);

// #ifdef DEBUG
//   // double norm;
//   printf(BLUE"");
//   gpu_vector_norm2(new_b, temp_vec5, half_vol, d_norm1);
//   qcuCudaMemcpy(&norm, d_norm1, sizeof(double), hipMemcpyDeviceToHost);
//   printf("norm2 of new_b is %lf\n", norm);
//   printf(CLR"");
// #endif

#ifdef DEBUG
  printf(RED"new odd b generated\n");
  printf(CLR"");
#endif



  if_end = odd_cg_inverter (odd_x, new_b, resid_vec, p_vec, \
                            temp_vec1, temp_vec2, temp_vec3, temp_vec4, temp_vec5, \
                            gauge, param, kappa, d_kappa, d_alpha, d_beta, \
                            d_denominator, d_numerator, d_coeff, d_norm1, d_norm2
  );

  if (!if_end) {
    printf("odd cg failed, donnot do even cg anymore, then exit\n");
    goto cg_end;
  }

  // odd succeed

  // even b
  generate_new_b_even (new_b, origin_even_b, odd_x,
                            gauge, d_kappa, d_coeff, param, kappa);

  if_end = even_cg_inverter (even_x, new_b, resid_vec, p_vec, \
                            temp_vec1, temp_vec2, temp_vec3, temp_vec4, temp_vec5, \
                            gauge, param, kappa, d_kappa, d_alpha, d_beta, \
                            d_denominator, d_numerator, d_coeff, d_norm1, d_norm2
  );
  if (!if_end) {
    printf("even cg failed, then exit\n");
    goto cg_end;
  }
cg_end:
  qcuCudaFree(temp_vec1);
  qcuCudaFree(temp_vec2);
  qcuCudaFree(temp_vec3);
  qcuCudaFree(temp_vec4);
  qcuCudaFree(temp_vec5);
  qcuCudaFree(p_vec);
  qcuCudaFree(resid_vec);
  qcuCudaFree(d_coeff);
  qcuCudaFree(d_kappa);

  qcuCudaFree(d_alpha);
  qcuCudaFree(d_beta);
  qcuCudaFree(d_denominator);
  qcuCudaFree(d_numerator);
  qcuCudaFree(d_norm1);
  qcuCudaFree(d_norm2);

  qcuCudaFree(new_b);
}