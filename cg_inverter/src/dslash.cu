#include <cstdio>
#include <cmath>
#include <assert.h>
#include <chrono>
#include <mpi.h>
#include "qcu.h"
#include <hip/hip_runtime.h>
#include "qcu_complex.cuh"
#include "qcu_dslash.cuh"
#include "qcu_macro.cuh"
#include "qcu_complex_computation.cuh"
#include "qcu_point.cuh"
#include "qcu_communicator.cuh"
#include "qcu_clover_dslash.cuh"
#include "qcu_wilson_dslash_neo.cuh"
#include "qcu_wilson_dslash.cuh"
#include "qcu_shift_storage.cuh"

#define qcuPrint() { \
    printf("function %s line %d...\n", __FUNCTION__, __LINE__); \
}


void* qcu_gauge;
void loadQcuGauge(void* gauge, QcuParam *param) {
  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];

  checkCudaErrors(hipMalloc(&qcu_gauge, sizeof(double) * Nd * Lx * Ly * Lz * Lt * (Nc-1) * Nc * 2));
  shiftGaugeStorage(qcu_gauge, gauge, TO_COALESCE, Lx, Ly, Lz, Lt);
}




void dslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity) {
  // parity ---- invert_flag

  // cloverDslashOneRound(fermion_out, fermion_in, gauge, param, 0);
  // cloverDslashOneRound(fermion_out, fermion_in, gauge, param, parity);
  // fullCloverDslashOneRound(fermion_out, fermion_in, gauge, param, 0);
  // wilsonDslashOneRound(fermion_out, fermion_in, gauge, param, parity);
  // callWilsonDslash(fermion_out, fermion_in, gauge, param, parity, 0);

  // callWilsonDslash(fermion_out, fermion_in, gauge, param, parity, 0);

  callWilsonDslashNaive(fermion_out, fermion_in, gauge, param, parity, 0);
}
void fullDslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int dagger_flag) {
  fullCloverDslashOneRound (fermion_out, fermion_in, gauge, param, dagger_flag);
}
