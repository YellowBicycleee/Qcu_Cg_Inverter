#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <assert.h>
#include <chrono>
#include <mpi.h>
#include "qcu.h"
#include <hip/hip_runtime.h>
#include "qcu_complex.cuh"
#include "qcu_complex_computation.cuh"
#include "qcu_macro.cuh"
#include "qcu_clover_dslash.cuh"
#include "qcu_communicator.cuh"
#include "qcu_wilson_dslash_neo.cuh"
#include "qcu_shift_storage_complex.cuh"
#define DEBUG
#define COALESCED_CG

extern MPICommunicator *mpi_comm;
extern void* qcu_gauge;
extern int process_rank;


// function pointers
void (*wilsonDslashFunction) (void *fermion_out, void *fermion_in, void *gauge, \
                              QcuParam *param, int parity, int dagger_flag);

void (*invertCloverDslashHalfFunction) (void *fermion_out, void *fermion_in, \
                              void *gauge, QcuParam *param, int parity);
void (*cloverVectorHalfFuntion) (void *fermion_out, void *fermion_in, \
                              void *gauge, QcuParam *param, int parity);



__attribute__((constructor)) void init_function() {
#ifdef COALESCED_CG
  wilsonDslashFunction = callWilsonDslashCoalesce;
  invertCloverDslashHalfFunction = invertCloverDslashHalfCoalesced;
  cloverVectorHalfFuntion = cloverVectorHalfCoalesced;
#else
  wilsonDslashFunction = callWilsonDslashNaive;
  invertCloverDslashHalfFunction = invertCloverDslashHalf;
  cloverVectorHalfFuntion = cloverVectorHalf;
#endif
}


/**
 * @brief clear the Complex vector named vec of vector_length elements to zero (kernel function)
 * 
 * @param vec 
 * @param vector_length 
 * @return void
 */
static __global__ void clearVectorKernel(void* vec, int vector_length) {
  int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
  int vol = blockDim.x * gridDim.x;
  Complex* src = static_cast<Complex*>(vec);

  if (thread_id >= vector_length) {
    return;
  }

  for (int i = thread_id; i < vector_length; i += vol) {
    src[i].clear2Zero();
  }
}

/**
 * @brief clear the Complex vector named vec of vector_length elements to zero (host function)
 * 
 * @param vec 
 * @param vector_length 
 * @return void
 */
static void clear_vector (void* vec, int vector_length) {
  int block_size = MAX_BLOCK_SIZE;
  int grid_size = (vector_length + block_size * Ns * Nc - 1) / (block_size * Ns * Nc);
  clearVectorKernel<<<grid_size, block_size>>>(vec, vector_length);
  qcuCudaDeviceSynchronize();
}


/**
 * @brief use this function to do one time MmV, when what to calc odd x
 * 
 * @param output_Ax result which means matrix A multiply vector x
 * @param input_x input vector x
 * @param temp_vec1 temporary space
 * @param temp_vec2 temporary space
 * @param gauge gauge
 * @param d_kappa kappa device pointer
 * @param param size of Lx Ly Lz Lt
 * @param dagger_flag 0 means no dagger, 1 means dagger
 * @param kappa kappa (double)
 */
void odd_matrix_mul_vector (void* output_Ax, void* input_x, void* temp_vec1, void* temp_vec2, void* gauge, void* d_kappa, QcuParam *param, int dagger_flag, double kappa) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  int parity;     // when use Doe, parity is odd which means 1, else 0
  Complex h_coeff;

  qcuCudaMemcpy(output_Ax, input_x, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);

  // part1 begin
  parity = 1;
  cloverVectorHalfFuntion (output_Ax, nullptr, gauge, param, parity);
  // part1 end

  // part2 begin
  parity = 0;
  wilsonDslashFunction(temp_vec1, input_x, gauge, param, parity, dagger_flag);
  parity = 0;
  invertCloverDslashHalfFunction (temp_vec1, nullptr, gauge, param, parity);  // clover invert
  parity = 1;
  wilsonDslashFunction(temp_vec2, temp_vec1, gauge, param, parity, dagger_flag);
  // part2 end

  h_coeff = Complex(-kappa * kappa, 0);
  qcuCudaMemcpy(d_kappa, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  // saxpy
  mpi_comm->interprocess_saxpy_barrier(temp_vec2, output_Ax, d_kappa, half_vol);  // coeff temp2 + x --->x
}

void full_odd_matrix_mul_vector (void* output_Ax, void* input_x, void* temp_vec1, void* temp_vec2, void* temp_vec3, void* gauge, void* d_kappa, QcuParam *param, double kappa) {

  int dagger_flag;
  dagger_flag = 0;
  odd_matrix_mul_vector (temp_vec3, input_x, temp_vec1, temp_vec2, gauge, \
                          d_kappa, param, dagger_flag, kappa);
  dagger_flag = 1;
  odd_matrix_mul_vector (output_Ax, temp_vec3, temp_vec1, temp_vec2, gauge, \
                          d_kappa, param, dagger_flag, kappa);
}
// current_b is temporary
bool if_even_converge(void* current_x, void* current_b_buffer, void* target_b, \
                    void* temp_vec1, void* temp_vec2, void* temp_vec3, \
                    void* gauge, void* d_kappa, void* d_coeff, \
                    void* d_norm1, void* d_norm2, QcuParam *param, double kappa \
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex h_coeff;
  double h_norm1; // norm(target_b)
  double h_norm2; // norm(target_b - current_b)

  int parity = 0;

  qcuCudaMemcpy(current_b_buffer, current_x, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);
  cloverVectorHalfFuntion (current_b_buffer, nullptr, gauge, param, parity);  // Ax ---> current_b_buffer

  //gpu_vector_norm2 (target_b, temp_vec3, half_vol, d_norm1);
  mpi_comm->interprocess_vector_norm(target_b, temp_vec3, half_vol, d_norm1);

  qcuCudaMemcpy (temp_vec2, target_b, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);     // target_b -----> temp_vec2
  h_coeff = Complex(-1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(current_b_buffer, temp_vec2, d_coeff, \
                half_vol); // temp_vec2 <--- target_b - current_b

  // gpu_vector_norm2(temp_vec2, temp_vec3, half_vol, d_norm2);
  mpi_comm->interprocess_vector_norm(temp_vec2, temp_vec3, half_vol, d_norm2);

  qcuCudaMemcpy(&h_norm1, d_norm1, sizeof(double), hipMemcpyDeviceToHost);
  qcuCudaMemcpy(&h_norm2, d_norm2, sizeof(double), hipMemcpyDeviceToHost);
#ifdef DEBUG
  printf("rank = %d, even difference :norm = %g, h_norm2 = %g, h_norm1=%g\n", process_rank, h_norm2 / h_norm1, h_norm2, h_norm1);
#endif
  return (h_norm2 / h_norm1 < 7e-15); // which means converge
}

// current_b is temporary
bool if_odd_converge(void* current_x, void* current_b_buffer, void* target_b, \
                    void* temp_vec1, void* temp_vec2, void* temp_vec3, \
                    void* gauge, void* d_kappa, void* d_coeff, \
                    void* d_norm1, void* d_norm2, QcuParam *param, double kappa
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex h_coeff;
  double h_norm1; // norm(target_b)
  double h_norm2; // norm(target_b - current_b)

  full_odd_matrix_mul_vector (current_b_buffer, current_x, \
                temp_vec1, temp_vec2, temp_vec3, gauge, d_kappa, param, kappa);

  // gpu_vector_norm2 (target_b, temp_vec3, half_vol, d_norm1);
  mpi_comm->interprocess_vector_norm (target_b, temp_vec3, half_vol, d_norm1);

  qcuCudaMemcpy (temp_vec2, target_b, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);     // target_b -----> temp_vec2
  h_coeff = Complex(-1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(current_b_buffer, temp_vec2, d_coeff, \
                half_vol); // temp_vec2 <--- target_b - current_b

  // gpu_vector_norm2(temp_vec2, temp_vec3, half_vol, d_norm2);
  mpi_comm->interprocess_vector_norm(temp_vec2, temp_vec3, half_vol, d_norm2);
  qcuCudaMemcpy(&h_norm1, d_norm1, sizeof(double), hipMemcpyDeviceToHost);
  qcuCudaMemcpy(&h_norm2, d_norm2, sizeof(double), hipMemcpyDeviceToHost);
#ifdef DEBUG
  // printf("difference %.64lf, \n h_norm1= %.64lf, \n h_norm2 = %.64lf\n", h_norm2 / h_norm1, h_norm1, h_norm2);
  // printf("difference %g\n", h_norm2 / h_norm1);
  printf("rank = %d, odd difference :norm = %g, h_norm2 = %g, h_norm1=%g\n", process_rank, h_norm2 / h_norm1, h_norm2, h_norm1);
#endif
  return (h_norm2 / h_norm1 < 1e-15); // which means converge
}

bool odd_cg_iter(void* iter_x_odd, void* target_b, void* resid_vec, void* p_vec, \
        void* temp_vec1, void* temp_vec2, void* temp_vec3, void* temp_vec4, void* temp_vec5, \
        void* gauge, QcuParam *param, double kappa, void* d_kappa, \
        void* d_alpha, void* d_beta, void* d_denominator, void* d_numerator, \
        void* d_coeff, void* d_norm1, void* d_norm2
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex alpha;
  Complex beta;
  Complex denominator;
  Complex numerator;
  Complex one(1,0);

  // <r, r>--->denominator
  mpi_comm->interprocess_inner_prod_barrier(resid_vec, resid_vec, \
                                    d_numerator, half_vol);  // <r, r> --> d_numerator


  full_odd_matrix_mul_vector (temp_vec4, p_vec, temp_vec1, \
                    temp_vec2, temp_vec3, gauge, d_kappa, param, kappa);
                    // Ap --->temp_vec4


  mpi_comm->interprocess_inner_prod_barrier(p_vec, temp_vec4, d_denominator, \
                    half_vol);  // <p, Ap> --> d_denominator

  qcuCudaMemcpy(&numerator, d_numerator, sizeof(Complex), hipMemcpyDeviceToHost);
  qcuCudaMemcpy(&denominator, d_denominator, sizeof(Complex), hipMemcpyDeviceToHost);
// #ifdef DEBUG
//   printf(RED"");
//   printf("numerator %lf %lf\n", numerator.real(), numerator.imag());
//   printf("denominator %lf %lf\n", denominator.real(), denominator.imag());
//   printf(CLR"");
// #endif

  alpha = numerator / denominator;
  qcuCudaMemcpy(d_alpha, &alpha, sizeof(Complex), hipMemcpyHostToDevice);

  mpi_comm->interprocess_saxpy_barrier(p_vec, iter_x_odd, d_alpha, half_vol); // x = x + \alpha p

  qcuCudaMemcpy(temp_vec1, resid_vec, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice); // copy r to temp_vec1  r'=r

  alpha = alpha * Complex(-1, 0);
  qcuCudaMemcpy(d_alpha, &alpha, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec4, temp_vec1, d_alpha, half_vol); // temp_vec4 = Ap, r'=r'-\alpha Ap------>temp_vec1

  if (if_odd_converge(iter_x_odd, temp_vec5, \
                      target_b, temp_vec2, temp_vec3,\
                      temp_vec4, gauge, d_kappa, d_coeff, \
                      d_norm1, d_norm2, param, kappa)
  ) { // donnot use temp_vec1 !!!!
    return true;
  }

  // <r, r> is in numerator
  mpi_comm->interprocess_inner_prod_barrier(temp_vec1, temp_vec1, \
                                            d_denominator, half_vol);  // <r', r'>

  qcuCudaMemcpy(&denominator, d_denominator, sizeof(Complex), hipMemcpyDeviceToHost);
  beta = denominator / numerator;
  qcuCudaMemcpy(d_beta, &beta, sizeof(Complex), hipMemcpyHostToDevice);
  // p = r' + \beta p
  gpu_sclar_multiply_vector (p_vec, d_beta, half_vol); // p_vec = \beta p_vec
  one = Complex(1, 0);
  qcuCudaMemcpy(d_coeff, &one, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec1, p_vec, d_coeff, half_vol); // p <-- r' + \beta p

  qcuCudaMemcpy(resid_vec, temp_vec1, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);  // r <--- r'

  return false;
}




bool even_cg_iter(void* iter_x_odd, void* target_b, void* resid_vec, void* p_vec, \
        void* temp_vec1, void* temp_vec2, void* temp_vec3, void* temp_vec4, void* temp_vec5, \
        void* gauge, QcuParam *param, double kappa, void* d_kappa, \
        void* d_alpha, void* d_beta, void* d_denominator, void* d_numerator, \
        void* d_coeff, void* d_norm1, void* d_norm2 \
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex alpha;
  Complex beta;
  Complex denominator;
  Complex numerator;
  Complex one(1,0);

  // <r, r>--->denominator
  mpi_comm->interprocess_inner_prod_barrier(resid_vec, resid_vec, \
                                    d_numerator, half_vol);  // <r, r> --> d_numerator

  qcuCudaMemcpy(temp_vec4, p_vec, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);
  cloverVectorHalfFuntion (temp_vec4, nullptr, gauge, param, 0);  // Ap --->temp_vec4


  mpi_comm->interprocess_inner_prod_barrier(p_vec, temp_vec4, d_denominator, \
                    half_vol);  // <p, Ap> --> d_denominator

  qcuCudaMemcpy(&numerator, d_numerator, sizeof(Complex), hipMemcpyDeviceToHost);
  qcuCudaMemcpy(&denominator, d_denominator, sizeof(Complex), hipMemcpyDeviceToHost);

  alpha = numerator / denominator;
  qcuCudaMemcpy(d_alpha, &alpha, sizeof(Complex), hipMemcpyHostToDevice);

  mpi_comm->interprocess_saxpy_barrier(p_vec, iter_x_odd, d_alpha, half_vol); // x = x + \alpha p

  qcuCudaMemcpy(temp_vec1, resid_vec, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice); // copy r to temp_vec1  r'=r

  alpha = alpha * Complex(-1, 0);
  qcuCudaMemcpy(d_alpha, &alpha, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec4, temp_vec1, d_alpha, half_vol); // temp_vec4 = Ap, r'=r-\alpha Ap------>temp_vec1

  if (if_even_converge(iter_x_odd, temp_vec5, \
                      target_b, temp_vec2, temp_vec3,\
                      temp_vec4, gauge, d_kappa, d_coeff, \
                      d_norm1, d_norm2, param, kappa)
  ) { // donnot use temp_vec1 !!!!
    return true;
  }

  // <r, r> is in numerator
  mpi_comm->interprocess_inner_prod_barrier(temp_vec1, temp_vec1, \
                                            d_denominator, half_vol);  // <r', r'>

  qcuCudaMemcpy(&denominator, d_denominator, sizeof(Complex), hipMemcpyDeviceToHost);
  beta = denominator / numerator;
  qcuCudaMemcpy(d_beta, &beta, sizeof(Complex), hipMemcpyHostToDevice);
  // p = r' + \beta p
  gpu_sclar_multiply_vector (p_vec, d_beta, half_vol); // p_vec = \beta p_vec
  one = Complex(1, 0);
  qcuCudaMemcpy(d_coeff, &one, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec1, p_vec, d_coeff, half_vol); // p <-- r' + \beta p

  qcuCudaMemcpy(resid_vec, temp_vec1, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);  // r <--- r'

  return false;
}


bool even_solver (void* iter_x_even, void* target_b, void* temp_vec, QcuParam *param) {
  bool if_converge;
  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;
  int parity;
  double difference;
  qcuCudaMemcpy(iter_x_even, target_b, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice);

  parity = 0;
  invertCloverDslashHalfFunction (iter_x_even, nullptr, nullptr, param, parity);
  return true;
}

// cg_even
bool even_cg_inverter (void* iter_x_even, void* target_b, void* resid_vec, void* p_vec,
  void* temp_vec1, void* temp_vec2, void* temp_vec3, void* temp_vec4, void* temp_vec5,\
  void* gauge, QcuParam *param, double kappa, void* d_kappa, \
  void* d_alpha, void* d_beta, void* d_denominator, void* d_numerator, void* d_coeff, void* d_norm1, void* d_norm2
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  int parity;
  bool if_converge = false;
  Complex h_coeff;

  clear_vector (iter_x_even, half_vol * Ns * Nc);  // x <-- 0
  // checkCudaErrors(hipMemset(iter_x_even, 0, sizeof(double) * 2 * half_vol * Ns * Nc));
  // b - Ax --->r
  qcuCudaMemcpy (resid_vec, target_b, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);      // r <-- b

  parity = 0;
  qcuCudaMemcpy (temp_vec1, iter_x_even, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);  // x-->temp_vec1
  cloverVectorHalfFuntion (temp_vec1, nullptr, gauge, param, parity);  // Ax ---> temp_vec1

  h_coeff = Complex(-1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec1, resid_vec, d_coeff, \
                                        half_vol);  // last: r <-- b-Ax

  if_converge =  if_even_converge(iter_x_even, temp_vec5, target_b, temp_vec1, \
                                  temp_vec2, temp_vec3, gauge, d_kappa, d_coeff, \
                                  d_norm1, d_norm2, param, kappa);
  if (if_converge) {
    return if_converge;
  }
  // then   r--->p
  qcuCudaMemcpy(p_vec, resid_vec, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);


  for (int i = 0; i < half_vol; i++) {
    if_converge = even_cg_iter(iter_x_even, target_b, resid_vec, p_vec, \
                          temp_vec1, temp_vec2, temp_vec3, temp_vec4, temp_vec5, \
                          gauge, param, kappa, d_kappa, d_alpha, d_beta, \
                          d_denominator, d_numerator, d_coeff, d_norm1, d_norm2);
    if (if_converge) {
      printf("even cg success! %d iterations\n", i+1);
      break;
    }
  }

  return if_converge;
}


// cg_odd
bool odd_cg_inverter (void* iter_x_odd, void* target_b, void* resid_vec, void* p_vec, \
  void* temp_vec1, void* temp_vec2, void* temp_vec3, void* temp_vec4, void* temp_vec5,\
  void* gauge, QcuParam *param, double kappa, void* d_kappa, \
  void* d_alpha, void* d_beta, void* d_denominator, void* d_numerator, void* d_coeff, void* d_norm1, void* d_norm2
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  bool converge = false;
  Complex h_coeff;

  clear_vector (iter_x_odd, half_vol * Ns * Nc);  // x <-- 0
  // checkCudaErrors(hipMemset(iter_x_odd, 0, sizeof(double) * 2 * half_vol * Ns * Nc));
  // b - Ax --->r
  qcuCudaMemcpy (resid_vec, target_b, sizeof(Complex) * half_vol * Ns * Nc, \
                hipMemcpyDeviceToDevice);      // r <-- b
  // second: Ax ---> temp_vec4
  full_odd_matrix_mul_vector (temp_vec4, iter_x_odd, temp_vec1, \
                            temp_vec2, temp_vec3, gauge, d_kappa, param, kappa);

  h_coeff = Complex(-1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(temp_vec4, resid_vec, d_coeff, \
                                        half_vol);  // last: r <-- b-Ax


  // If converge return x
  if (if_odd_converge(iter_x_odd, temp_vec4, target_b, temp_vec1, temp_vec2, temp_vec3, gauge, d_kappa, d_coeff, d_norm1, d_norm2, param, kappa)) {
    printf("cg suceess!\n");
    // goto odd_cg_end;
    return converge;
  }

#ifdef DEBUG
  printf(RED"first iteration passed\n");
  printf(CLR"");
#endif
  // p <-- r
  qcuCudaMemcpy(p_vec, resid_vec, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice);


  for (int i = 0; i < half_vol; i++) {
#ifdef DEBUG
    printf(RED"iteration %d", i+1);
    printf(CLR"");
#endif


  // test inner prod
// #ifdef DEBUG
//   Complex temp;
//   mpi_comm->interprocess_inner_prod_barrier(target_b, target_b, temp_vec5, half_vol);
//   qcuCudaMemcpy(&temp, temp_vec5, sizeof(Complex), hipMemcpyDeviceToHost);
//   printf(BLUE"temp.real = %lf, temp.imag = %lf\n", temp.real(), temp.imag());
//   printf(CLR"");
// #endif

    converge = odd_cg_iter(iter_x_odd, target_b, resid_vec, p_vec, \
                          temp_vec1, temp_vec2, temp_vec3, temp_vec4, temp_vec5, \
                          gauge, param, kappa, d_kappa, d_alpha, d_beta, \
                          d_denominator, d_numerator, d_coeff, d_norm1, d_norm2);

    if (converge) {
      printf("odd cg success! %d iterations\n", i+1);
      break;
    }
  }


// odd_cg_end:
  return converge;
}

void generate_new_b_even (void* new_even_b, void* origin_even_b, void* res_odd_x, \
                          void* gauge, void* d_kappa, void* d_coeff, \
                        QcuParam *param, double kappa
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  int parity = 0;
  int dagger_flag = 0;
  Complex h_kappa(kappa, 0);
  Complex h_coeff;
  qcuCudaMemcpy(d_kappa, &h_kappa, sizeof(Complex), hipMemcpyHostToDevice);

  // D_{eo}x_{o} ----> new_even_b
  wilsonDslashFunction (new_even_b, res_odd_x, gauge, param, parity, dagger_flag);
  // kappa D_{eo}x_{o} ----> new_even_b
  mpi_comm->interprocess_sax_barrier (new_even_b, d_kappa, half_vol);

  h_coeff = Complex(1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  // kappa D_{eo}x_{o} + even_b ----> new_even_b
  mpi_comm->interprocess_saxpy_barrier(origin_even_b, new_even_b, d_coeff, half_vol);
}


// modify b, half-length vector
void generate_new_b_odd(void* new_b, void* origin_odd_b, void* origin_even_b, \
                        void* temp_vec1, void* temp_vec2, void* temp_vec3, \
                        void* gauge, void* d_kappa, void* d_coeff, \
                        QcuParam *param, double kappa
) {

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

  Complex h_kappa;
  Complex h_coeff;
  int parity;
  int dagger_flag;

  // even b ----> temp_vec1
  qcuCudaMemcpy(temp_vec1, origin_even_b, sizeof(Complex) * half_vol * Ns * Nc, hipMemcpyDeviceToDevice);

  parity = 0;
  invertCloverDslashHalfFunction (temp_vec1, nullptr, gauge, param, parity); // A^{-1}_{ee}b_{e} ---> temp_vec1

  parity = 1;
  dagger_flag = 0;
  wilsonDslashFunction (new_b, temp_vec1, gauge, param, parity, dagger_flag); //  D_{oe}A^{-1}_{ee}b_{e} ----> new_b

  // kappa D_{oe}A^{-1}_{ee}b_{e}
  h_kappa = Complex(kappa, 0);
  qcuCudaMemcpy(d_kappa, &h_kappa, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_sax_barrier (new_b, d_kappa, half_vol);

  h_coeff = Complex(1, 0);
  qcuCudaMemcpy(d_coeff, &h_coeff, sizeof(Complex), hipMemcpyHostToDevice);
  mpi_comm->interprocess_saxpy_barrier(origin_odd_b, new_b, d_coeff, half_vol);
}


void cg_inverter(void* b_vector, void* x_vector, void *gauge, QcuParam *param) {
  double kappa = 0.125;

  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol >> 1;

#ifdef DEBUG
  printf("begin func cg, begin .....\n");
#endif


#ifdef COALESCED_CG
  gauge = qcu_gauge;
  void *origin_x_vector = x_vector;
  void *coalesced_b_vector;
  void *coalesced_x_vector;

  qcuCudaMalloc(&coalesced_b_vector, sizeof(Complex) * vol * Ns * Nc);
  qcuCudaMalloc(&coalesced_x_vector, sizeof(Complex) * vol * Ns * Nc);
  void* origin_vector_eo = b_vector;
  void* coalesced_vector_eo = coalesced_b_vector;
  shiftVectorStorageTwoDouble(coalesced_vector_eo, origin_vector_eo, TO_COALESCE, Lx, Ly, Lz, Lt);
  origin_vector_eo = static_cast<void*>(static_cast<Complex*>(b_vector) + half_vol * Ns * Nc);
  coalesced_vector_eo = static_cast<void*>(static_cast<Complex*>(coalesced_b_vector) + half_vol * Ns * Nc);;
  shiftVectorStorageTwoDouble(coalesced_vector_eo, origin_vector_eo, TO_COALESCE, Lx, Ly, Lz, Lt);

  x_vector = coalesced_x_vector;
  b_vector = coalesced_b_vector;
#endif


  bool if_end = false;
  // ptrs doesn't need new memory
  void* origin_even_b;
  void* origin_odd_b;
  void* even_x;
  void* odd_x;

  // ptrs need to allocate memory
  void* temp_vec1;
  void* temp_vec2;
  void* temp_vec3;
  void* temp_vec4;
  void* temp_vec5;

  void* p_vec;
  void* resid_vec;

  void* d_coeff;
  void* d_kappa;
  void* d_alpha;
  void* d_beta;
  void* d_denominator;
  void* d_numerator;
  void* d_norm1;
  void* d_norm2;
  void* new_b;

  int dagger_flag;

  origin_even_b = b_vector;
  origin_odd_b = static_cast<void*>(static_cast<Complex*>(b_vector) \
                  + half_vol * Ns * Nc);
  even_x = x_vector;
  odd_x = static_cast<void*>(static_cast<Complex*>(x_vector) + half_vol * Ns * Nc);

  // memory allocation
  qcuCudaMalloc(&temp_vec1, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&temp_vec2, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&temp_vec3, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&temp_vec4, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&temp_vec5, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&p_vec, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&resid_vec, sizeof(Complex) * half_vol * Ns * Nc);
  qcuCudaMalloc(&d_coeff, sizeof(Complex));
  qcuCudaMalloc(&d_kappa, sizeof(Complex));

  qcuCudaMalloc(&d_alpha, sizeof(Complex));
  qcuCudaMalloc(&d_beta, sizeof(Complex));
  qcuCudaMalloc(&d_denominator, sizeof(Complex));
  qcuCudaMalloc(&d_numerator, sizeof(Complex));
  qcuCudaMalloc(&d_norm1, sizeof(Complex));
  qcuCudaMalloc(&d_norm2, sizeof(Complex));

  qcuCudaMalloc(&new_b, sizeof(Complex) * half_vol * Ns * Nc);



#ifdef DEBUG
  printf("memory allocated, begin .....\n");
#endif
  // odd new_b
  generate_new_b_odd(temp_vec3, origin_odd_b, origin_even_b, temp_vec1, \
                    temp_vec2, temp_vec4, gauge, d_kappa, d_coeff, param, kappa);
#ifdef DEBUG
  printf("odd new_b generated, begin .....\n");
#endif
  // odd dagger D new_b
  dagger_flag = 1;
  odd_matrix_mul_vector (new_b, temp_vec3, temp_vec1, temp_vec2, \
                         gauge, d_kappa, param, dagger_flag, kappa);

  if_end = odd_cg_inverter (odd_x, new_b, resid_vec, p_vec, \
                            temp_vec1, temp_vec2, temp_vec3, temp_vec4, temp_vec5, \
                            gauge, param, kappa, d_kappa, d_alpha, d_beta, \
                            d_denominator, d_numerator, d_coeff, d_norm1, d_norm2);

  if (!if_end) {
    printf("odd cg failed, donnot do even cg anymore, then exit\n");
    goto cg_end;
  }


  // even b
  generate_new_b_even (new_b, origin_even_b, odd_x,
                            gauge, d_kappa, d_coeff, param, kappa);
  // TODO:
  // if_end = even_cg_inverter (even_x, new_b, resid_vec, p_vec, \
  //                           temp_vec1, temp_vec2, temp_vec3, temp_vec4, temp_vec5, \
  //                           gauge, param, kappa, d_kappa, d_alpha, d_beta, \
  //                           d_denominator, d_numerator, d_coeff, d_norm1, d_norm2
  // );

  if_end = even_solver(even_x, new_b, temp_vec1, param);
  if (!if_end) {
    printf("even cg failed, then exit\n");
    goto cg_end;
  }
cg_end:
  qcuCudaFree(temp_vec1);
  qcuCudaFree(temp_vec2);
  qcuCudaFree(temp_vec3);
  qcuCudaFree(temp_vec4);
  qcuCudaFree(temp_vec5);
  qcuCudaFree(p_vec);
  qcuCudaFree(resid_vec);
  qcuCudaFree(d_coeff);
  qcuCudaFree(d_kappa);

  qcuCudaFree(d_alpha);
  qcuCudaFree(d_beta);
  qcuCudaFree(d_denominator);
  qcuCudaFree(d_numerator);
  qcuCudaFree(d_norm1);
  qcuCudaFree(d_norm2);

  qcuCudaFree(new_b);

#ifdef COALESCED_CG
  x_vector = origin_x_vector;

  origin_vector_eo = x_vector;
  coalesced_vector_eo = coalesced_x_vector;
  shiftVectorStorageTwoDouble(origin_vector_eo, coalesced_vector_eo, TO_NON_COALESCE, Lx, Ly, Lz, Lt);
  origin_vector_eo = static_cast<void*>(static_cast<Complex*>(x_vector) + half_vol * Ns * Nc);
  coalesced_vector_eo = static_cast<void*>(static_cast<Complex*>(coalesced_x_vector) + half_vol * Ns * Nc);
  shiftVectorStorageTwoDouble(origin_vector_eo, coalesced_vector_eo, TO_NON_COALESCE, Lx, Ly, Lz, Lt);
  //   shiftVectorStorageTwoDouble(fermion_out, coalesced_fermion_out, TO_NON_COALESCE, Lx, Ly, Lz, Lt);
  qcuCudaFree(coalesced_b_vector);
  qcuCudaFree(coalesced_x_vector);
#endif
}

