#include "hip/hip_runtime.h"
// #include "qcu_wilson_dslash_neo.cuh"
#include "qcu_complex.cuh"
#include "qcu_point.cuh"
#include "qcu_communicator.cuh"
// #include "qcu_shift_storage.cuh"
#include <chrono>
#include "qcu_wilson_dslash_new_new.cuh"
#define INCLUDE_COMPUTATION




extern int grid_x;
extern int grid_y;
extern int grid_z;
extern int grid_t;
extern MPICommunicator *mpi_comm;

// static void* coalesced_fermion_in;
// static void* coalesced_fermion_out;
// static void* coalesced_gauge;
// extern void* qcu_gauge;



// // assumption: any [0][1] of pointer is accessible
// template<typename T>
// __device__ __host__ inline void complex_mul(T* result, T* left, T* right){
//   res[0] = left[0] * left[1] - right[0] * right[1];
//   res[1] = left[0] * right[1] + left[1] * right[0];
// }
// template<typename T>
// __device__ __host__ inline void complex_add(T* result, T* left, T* right){
//   result[0] = left[0] + right[0];
//   result[1] = left[1] + right[1];
// }
// template<typename T>
// __device__ __host__ inline void complex_sub(T* result, T* left, T* right){
//   result[0] = left[0] - right[0];
//   result[1] = left[1] - right[1];
// }
// template<typename T>
// __device__ __host__ inline void complex_mul_i(T* result, T* src) {
//   result[0] = -src[1];
//   result[1] = src[0];
// }
// template<typename T>
// __device__ __host__ inline void complex_self_mul_i(T* result) {
//   T temp = result[0];
//   result[0] = -result[1];
//   result[1] = temp;
// }

// template<typename T>
// __device__ __host__ inline void complex_mul_minus_i(T* result, T* src) {
//   result[0] = src[1];
//   result[1] = -src[0];
// }
// template<typename T>
// __device__ __host__ inline void complex_self_mul_minus_i(T* result, T* src) {
//   T temp = result[0];
//   result[0] = result[1];
//   result[1] = -temp;
// }

// template<typename T>
// __device__ __host__ inline void complex_clear(T* result) {
//   result[0] = 0;
//   result[1] = 0;
// }
// template <typename T>
// __device__ __host__ inline void complex_assign(T* result, T* src) {
//   result[0] = src[0];
//   result[1] = src[1];
// }
// template <typename T>
// __device__ __host__ inline void complex_sub_assign(T* result, T* src) {
//   result[0] -= src[0];
//   result[1] -= src[1];
// }
// template <typename T>
// __device__ __host__ inline void complex_add_assign(T* result, T* src) {
//   result[0] += src[0];
//   result[1] += src[1];
// }


// template <typename T>
// __device__ __host__ inline void complex_self_multiply_complex(T* result, T* src) {
//   T real = src[0] * src[1] - result[0] * result[1];
//   T imag = src[0] * result[1] + src[1] * result[0];
//   result[0] = real;
//   result[1] = imag;
// }
// template <typename T>
// __device__ __host__ inline void complex_self_multiply_double(T* result, T* src) {
//   result[0] = result[0] * src[0];
//   result[1] = result[1] * src[1];
// }






static __device__ inline void reconstructSU3(double *su3)
{
  // su3[6] = (su3[1] * su3[5] - su3[2] * su3[4]).conj();
  // su3[7] = (su3[2] * su3[3] - su3[0] * su3[5]).conj();
  // su3[8] = (su3[0] * su3[4] - su3[1] * su3[3]).conj();
  su3[6 * 2 + 0] = (su3[1 * 2 + 0] * su3[5 * 2 + 0] - su3[1 * 2 + 1] * su3[5 * 2 + 1]) \
                 - (su3[2 * 2 + 0] * su3[4 * 2 + 0] - su3[2 * 2 + 1] * su3[4 * 2 + 1]);
  su3[6 * 2 + 1] = (su3[2 * 2 + 1] * su3[4 * 2 + 0] + su3[2 * 2 + 0] * su3[4 * 2 + 1]) \
                 - (su3[1 * 2 + 1] * su3[5 * 2 + 0] + su3[1 * 2 + 0] * su3[5 * 2 + 1]); // conj()

  su3[7 * 2 + 0] = (su3[2 * 2 + 0] * su3[3 * 2 + 0] - su3[2 * 2 + 1] * su3[3 * 2 + 1]) \
                 - (su3[0 * 2 + 0] * su3[5 * 2 + 0] - su3[0 * 2 + 1] * su3[5 * 2 + 1]);
  su3[7 * 2 + 1] = (su3[0 * 2 + 1] * su3[5 * 2 + 0] + su3[0 * 2 + 0] * su3[5 * 2 + 1]) \
                 - (su3[2 * 2 + 1] * su3[3 * 2 + 0] + su3[2 * 2 + 0] * su3[3 * 2 + 1]); // conj()

  su3[8 * 2 + 0] = (su3[0 * 2 + 0] * su3[4 * 2 + 0] - su3[0 * 2 + 1] * su3[4 * 2 + 1]) \
                 - (su3[1 * 2 + 0] * su3[3 * 2 + 0] - su3[1 * 2 + 1] * su3[3 * 2 + 1]);
  su3[8 * 2 + 1] = (su3[1 * 2 + 1] * su3[3 * 2 + 0] + su3[1 * 2 + 0] * su3[3 * 2 + 1]) \
                 - (su3[0 * 2 + 1] * su3[4 * 2 + 0] + su3[0 * 2 + 0] * su3[4 * 2 + 1]); // conj()
}



__device__ inline void loadGauge(double* u_local, void* gauge_ptr, int direction, const Point& p, int Lx, int Ly, int Lz, int Lt) {
  double* u = reinterpret_cast<double*>(p.getPointGauge(static_cast<Complex*>(gauge_ptr), direction, Lx, Ly, Lz, Lt));
  for (int i = 0; i < (Nc - 1) * Nc * 2; i++) {
    u_local[i] = u[i];
  }
  // reconstructSU3(u_local);
  reconstructSU3(reinterpret_cast<double*>(u_local));
}





__device__ inline void loadVector(double* src_local, void* fermion_in, const Point& p, int Lx, int Ly, int Lz, int Lt) {
  // Complex* src = p.getPointVector(static_cast<Complex *>(fermion_in), Lx, Ly, Lz, Lt);
  double* src_double = reinterpret_cast<double*>(p.getPointVector(static_cast<Complex *>(fermion_in), Lx, Ly, Lz, Lt));

  for (int i = 0; i < Ns * Nc * 2; i++) {
    src_local[i] = src_double[i];
  }
}



static __global__ void mpiDslashNew(void *gauge, void *fermion_in, void *fermion_out,int Lx, int Ly, int Lz, int Lt, int parity, int grid_x, int grid_y, int grid_z, int grid_t, double flag_param) {
  assert(parity == 0 || parity == 1);
  // __shared__ double shared_buffer[BLOCK_SIZE * Ns * Nc * 2];

  Lx >>= 1;

  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int t = thread_id / (Lz * Ly * Lx);
  int z = thread_id % (Lz * Ly * Lx) / (Ly * Lx);
  int y = thread_id % (Ly * Lx) / Lx;
  int x = thread_id % Lx;

  int coord_boundary;
  double flag = flag_param;


  Point p(x, y, z, t, parity);
  Point move_point;
  Complex u_local[Nc * Nc];   // for GPU
  Complex src_local[Ns * Nc]; // for GPU
  Complex dst_local[Ns * Nc]; // for GPU

  double* u_local_double_ptr = reinterpret_cast<double*>(u_local);
  double* src_local_double_ptr = reinterpret_cast<double*>(src_local);
  double* dst_local_double_ptr = reinterpret_cast<double*>(dst_local);
  // Complex temp;
  // Complex temp1;
  // Complex temp2;

  double temp_reg[2];
  double temp_res1[2];
  double temp_res2[2];
  int eo = (y+z+t) & 0x01;

  for (int i = 0; i < Ns * Nc; i++) {
    dst_local[i].clear2Zero();
  }

  // \mu = 1
  loadGauge(reinterpret_cast<double*>(u_local), gauge, X_DIRECTION, p, Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, 0, Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, Lx, Ly, Lz, Lt);
  // x front    x == Lx-1 && parity != eo
  coord_boundary = (grid_x > 1 && x == Lx-1 && parity != eo) ? Lx-1 : Lx;
  if (x < coord_boundary) {

#ifdef INCLUDE_COMPUTATION
#pragma unroll
    for (int i = 0; i < Nc; i++) {
      // temp1.clear2Zero();
      // temp2.clear2Zero();
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
#pragma unroll
      for (int j = 0; j < Nc; j++) {
        // temp1 += (src_local[0 * Nc + j] - src_local[3 * Nc + j].multipy_i() * flag) * u_local[i * Nc + j];
        temp_reg[0] = (src_local_double_ptr[(0 * Nc + j) * 2 + 0] \
                    - (-src_local_double_ptr[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local_double_ptr[(0 * Nc + j) * 2 + 1] \
                    - (src_local_double_ptr[(3 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 0];

        // second row vector with col vector
        // temp2 += (src_local[1 * Nc + j] - src_local[2 * Nc + j].multipy_i() * flag) * u_local[i * Nc + j];
        temp_reg[0] = (src_local_double_ptr[(1 * Nc + j) * 2 + 0] \
                    - (-src_local_double_ptr[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local_double_ptr[(1 * Nc + j) * 2 + 1] \
                    - (src_local_double_ptr[(2 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 0];
      }
      // dst_local[0 * Nc + i] += temp1;
      // dst_local[3 * Nc + i] += temp1.multipy_i() * flag;
      dst_local_double_ptr[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local_double_ptr[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 0] += -flag * temp_res1[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 1] += flag * temp_res1[0];

      // dst_local[1 * Nc + i] += temp2;
      // dst_local[2 * Nc + i] += temp2.multipy_i() * flag;
      dst_local_double_ptr[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local_double_ptr[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 0] += -flag * temp_res2[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 1] += flag * temp_res2[0];
    }
#endif
  }
  // x back   x==0 && parity == eo
  move_point = p.move(BACK, 0, Lx, Ly, Lz, Lt);
  loadGauge(reinterpret_cast<double*>(u_local), gauge, X_DIRECTION, move_point, Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, Lx, Ly, Lz, Lt);;

  coord_boundary = (grid_x > 1 && x==0 && parity == eo) ? 1 : 0;
  if (x >= coord_boundary) {
#ifdef INCLUDE_COMPUTATION
#pragma unroll
    for (int i = 0; i < Nc; i++) {
      // temp1.clear2Zero();
      // temp2.clear2Zero();
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
#pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        // temp1 += (src_local[0 * Nc + j] + src_local[3 * Nc + j].multipy_i() * flag) *
        //       u_local[j * Nc + i].conj(); // transpose and conj

        temp_reg[0] = (src_local_double_ptr[(0 * Nc + j) * 2 + 0] \
                    + (-src_local_double_ptr[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local_double_ptr[(0 * Nc + j) * 2 + 1] \
                    + (src_local_double_ptr[(3 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local_double_ptr[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local_double_ptr[(j * Nc + i) * 2 + 0];


        // second row vector with col vector
        // temp2 += (src_local[1 * Nc + j] + src_local[2 * Nc + j].multipy_i() * flag) *
        //       u_local[j * Nc + i].conj(); // transpose and conj
        temp_reg[0] = (src_local_double_ptr[(1 * Nc + j) * 2 + 0] \
                    + (-src_local_double_ptr[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local_double_ptr[(1 * Nc + j) * 2 + 1] \
                    + (src_local_double_ptr[(2 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local_double_ptr[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local_double_ptr[(j * Nc + i) * 2 + 0];
      }
      // dst_local[0 * Nc + i] += temp1;
      // dst_local[3 * Nc + i] += temp1.multipy_minus_i() * flag;
      dst_local_double_ptr[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local_double_ptr[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 0] += flag * temp_res1[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 1] += -flag * temp_res1[0];

      // dst_local[1 * Nc + i] += temp2;
      // dst_local[2 * Nc + i] += temp2.multipy_minus_i() * flag;
      dst_local_double_ptr[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local_double_ptr[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 0] += flag * temp_res2[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 1] += -flag * temp_res2[0];
    }
#endif
  }

  // \mu = 2
  // y front
  loadGauge(reinterpret_cast<double*>(u_local), gauge, Y_DIRECTION, p, Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, 1, Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, Lx, Ly, Lz, Lt);

  coord_boundary = (grid_y > 1) ? Ly-1 : Ly;
  if (y < coord_boundary) {
#ifdef INCLUDE_COMPUTATION
#pragma unroll
    for (int i = 0; i < Nc; i++) {
      // temp1.clear2Zero();
      // temp2.clear2Zero();
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
#pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        // temp1 += (src_local[0 * Nc + j] + src_local[3 * Nc + j] * flag) * u_local[i * Nc + j];
        temp_reg[0] = (src_local_double_ptr[(0 * Nc + j) * 2 + 0] \
                    + (src_local_double_ptr[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local_double_ptr[(0 * Nc + j) * 2 + 1] \
                    + (src_local_double_ptr[(3 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 0];


        // second row vector with col vector
        // temp2 += (src_local[1 * Nc + j] - src_local[2 * Nc + j] *  flag) * u_local[i * Nc + j];
        temp_reg[0] = (src_local_double_ptr[(1 * Nc + j) * 2 + 0] \
                    - (src_local_double_ptr[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local_double_ptr[(1 * Nc + j) * 2 + 1] \
                    - (src_local_double_ptr[(2 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 0];

      }
      // dst_local[0 * Nc + i] += temp1;
      // dst_local[3 * Nc + i] += temp1 * flag;
      dst_local_double_ptr[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local_double_ptr[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 0] += flag * temp_res1[0];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 1] += flag * temp_res1[1];

      // dst_local[1 * Nc + i] += temp2;
      // dst_local[2 * Nc + i] += -temp2 * flag;
      dst_local_double_ptr[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local_double_ptr[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 0] += -flag * temp_res2[0];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 1] += -flag * temp_res2[1];
    }
#endif
  }

  // y back
  move_point = p.move(BACK, 1, Lx, Ly, Lz, Lt);
  loadGauge(reinterpret_cast<double*>(u_local), gauge, Y_DIRECTION, move_point, Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, Lx, Ly, Lz, Lt);

  coord_boundary = (grid_y > 1) ? 1 : 0;
  if (y >= coord_boundary) {
#ifdef INCLUDE_COMPUTATION
#pragma unroll
    for (int i = 0; i < Nc; i++) {
      // temp1.clear2Zero();
      // temp2.clear2Zero();
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
#pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        // temp1 += (src_local[0 * Nc + j] - src_local[3 * Nc + j] * flag) * u_local[j * Nc + i].conj(); // transpose and conj
        temp_reg[0] = (src_local_double_ptr[(0 * Nc + j) * 2 + 0] \
                    - (src_local_double_ptr[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local_double_ptr[(0 * Nc + j) * 2 + 1] \
                    - (src_local_double_ptr[(3 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local_double_ptr[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local_double_ptr[(j * Nc + i) * 2 + 0];

        // second row vector with col vector
        // temp2 += (src_local[1 * Nc + j] + src_local[2 * Nc + j] * flag) * u_local[j * Nc + i].conj(); // transpose and conj
        temp_reg[0] = (src_local_double_ptr[(1 * Nc + j) * 2 + 0] \
                    + (src_local_double_ptr[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local_double_ptr[(1 * Nc + j) * 2 + 1] \
                    + (src_local_double_ptr[(2 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local_double_ptr[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local_double_ptr[(j * Nc + i) * 2 + 0];

      }
      // dst_local[0 * Nc + i] += temp1;
      // dst_local[3 * Nc + i] += -temp1 * flag;
      dst_local_double_ptr[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local_double_ptr[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 0] += -flag * temp_res1[0];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 1] += -flag * temp_res1[1];

      // dst_local[1 * Nc + i] += temp2;
      // dst_local[2 * Nc + i] += temp2 * flag;
      dst_local_double_ptr[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local_double_ptr[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 0] += flag * temp_res2[0];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 1] += flag * temp_res2[1];
    }
#endif
  }

  // \mu = 3
  // z front
  loadGauge(reinterpret_cast<double*>(u_local), gauge, Z_DIRECTION, p, Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, 2, Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, Lx, Ly, Lz, Lt);
  coord_boundary = (grid_z > 1) ? Lz-1 : Lz;
  if (z < coord_boundary) {
#ifdef INCLUDE_COMPUTATION
#pragma unroll
    for (int i = 0; i < Nc; i++) {
      // temp1.clear2Zero();
      // temp2.clear2Zero();
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
#pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        // temp1 += (src_local[0 * Nc + j] - src_local[2 * Nc + j].multipy_i() * flag) * u_local[i * Nc + j];
        temp_reg[0] = (src_local_double_ptr[(0 * Nc + j) * 2 + 0] \
                    - (-src_local_double_ptr[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local_double_ptr[(0 * Nc + j) * 2 + 1] \
                    - (src_local_double_ptr[(2 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 0];

        // second row vector with col vector
        // temp2 += (src_local[1 * Nc + j] + src_local[3 * Nc + j].multipy_i() * flag) * u_local[i * Nc + j];
        temp_reg[0] = (src_local_double_ptr[(1 * Nc + j) * 2 + 0] \
                    + (-src_local_double_ptr[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local_double_ptr[(1 * Nc + j) * 2 + 1] \
                    + (src_local_double_ptr[(3 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 0];
      }
      // dst_local[0 * Nc + i] += temp1;
      // dst_local[2 * Nc + i] += temp1.multipy_i() * flag;
      dst_local_double_ptr[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local_double_ptr[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 0] += -flag * temp_res1[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 1] += flag * temp_res1[0];

      // dst_local[1 * Nc + i] += temp2;
      // dst_local[3 * Nc + i] += temp2.multipy_minus_i() * flag;
      dst_local_double_ptr[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local_double_ptr[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 0] += flag * temp_res2[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 1] += -flag * temp_res2[0];
    }
#endif
  }

  // z back
  move_point = p.move(BACK, 2, Lx, Ly, Lz, Lt);
  loadGauge(reinterpret_cast<double*>(u_local), gauge, Z_DIRECTION, move_point, Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, Lx, Ly, Lz, Lt);

  coord_boundary = (grid_z > 1) ? 1 : 0;
  if (z >= coord_boundary) {
#ifdef INCLUDE_COMPUTATION
#pragma unroll
    for (int i = 0; i < Nc; i++) {
      // temp1.clear2Zero();
      // temp2.clear2Zero();
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
#pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        // temp1 += (src_local[0 * Nc + j] + src_local[2 * Nc + j].multipy_i() * flag) *
              // u_local[j * Nc + i].conj(); // transpose and conj
        temp_reg[0] = (src_local_double_ptr[(0 * Nc + j) * 2 + 0] \
                    + (-src_local_double_ptr[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local_double_ptr[(0 * Nc + j) * 2 + 1] \
                    + (src_local_double_ptr[(2 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local_double_ptr[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local_double_ptr[(j * Nc + i) * 2 + 0];


        // second row vector with col vector
        // temp2 += (src_local[1 * Nc + j] - src_local[3 * Nc + j].multipy_i() * flag) *
        //       u_local[j * Nc + i].conj(); // transpose and conj
        temp_reg[0] = (src_local_double_ptr[(1 * Nc + j) * 2 + 0] \
                    - (-src_local_double_ptr[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local_double_ptr[(1 * Nc + j) * 2 + 1] \
                    - (src_local_double_ptr[(3 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local_double_ptr[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local_double_ptr[(j * Nc + i) * 2 + 0];
      }
      // dst_local[0 * Nc + i] += temp1;
      // dst_local[2 * Nc + i] += temp1.multipy_minus_i() * flag;
      dst_local_double_ptr[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local_double_ptr[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 0] += flag * temp_res1[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 1] += -flag * temp_res1[0];

      // dst_local[1 * Nc + i] += temp2;
      // dst_local[3 * Nc + i] += temp2.multipy_i() * flag;
      dst_local_double_ptr[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local_double_ptr[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 0] += -flag * temp_res2[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 1] += flag * temp_res2[0];
    }
#endif
  }

  // t: front
  // loadGauge(u_local, gauge, 3, p, Lx, Ly, Lz, Lt);
  loadGauge(reinterpret_cast<double*>(u_local), gauge, T_DIRECTION, p, Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, 3, Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, Lx, Ly, Lz, Lt);

  coord_boundary = (grid_t > 1) ? Lt-1 : Lt;
  if (t < coord_boundary) {
#ifdef INCLUDE_COMPUTATION
#pragma unroll
    for (int i = 0; i < Nc; i++) {
      // temp1.clear2Zero();
      // temp2.clear2Zero();
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
#pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        // temp1 += (src_local[0 * Nc + j] - src_local[2 * Nc + j] * flag) * u_local[i * Nc + j];
        temp_reg[0] = (src_local_double_ptr[(0 * Nc + j) * 2 + 0] \
                    - (src_local_double_ptr[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local_double_ptr[(0 * Nc + j) * 2 + 1] \
                    - (src_local_double_ptr[(2 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 0];
        // second row vector with col vector
        // temp2 += (src_local[1 * Nc + j] - src_local[3 * Nc + j] * flag) * u_local[i * Nc + j];
        temp_reg[0] = (src_local_double_ptr[(1 * Nc + j) * 2 + 0] \
                    - (src_local_double_ptr[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local_double_ptr[(1 * Nc + j) * 2 + 1] \
                    - (src_local_double_ptr[(3 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local_double_ptr[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local_double_ptr[(i * Nc + j) * 2 + 0];
      }
      // dst_local[0 * Nc + i] += temp1;
      // dst_local[2 * Nc + i] += -temp1 * flag;
      dst_local_double_ptr[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local_double_ptr[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 0] += -flag * temp_res1[0];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 1] += -flag * temp_res1[1];

      // dst_local[1 * Nc + i] += temp2;
      // dst_local[3 * Nc + i] += -temp2 * flag;
      dst_local_double_ptr[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local_double_ptr[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 0] += -flag * temp_res2[0];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 1] += -flag * temp_res2[1];
    }
#endif
  }
  // t: back
  move_point = p.move(BACK, 3, Lx, Ly, Lz, Lt);
  loadGauge(reinterpret_cast<double*>(u_local), gauge, T_DIRECTION, move_point, Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, Lx, Ly, Lz, Lt);

  coord_boundary = (grid_t > 1) ? 1 : 0;
  if (t >= coord_boundary) {
#ifdef INCLUDE_COMPUTATION
#pragma unroll
    for (int i = 0; i < Nc; i++) {
      // temp1.clear2Zero();
      // temp2.clear2Zero();
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
#pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        // temp1 += (src_local[0 * Nc + j] + src_local[2 * Nc + j] * flag) * u_local[j * Nc + i].conj(); // transpose and conj
        temp_reg[0] = (src_local_double_ptr[(0 * Nc + j) * 2 + 0] \
                    + (src_local_double_ptr[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local_double_ptr[(0 * Nc + j) * 2 + 1] \
                    + (src_local_double_ptr[(2 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local_double_ptr[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local_double_ptr[(j * Nc + i) * 2 + 0];

        // second row vector with col vector
        // temp2 += (src_local[1 * Nc + j] + src_local[3 * Nc + j] * flag) * u_local[j * Nc + i].conj(); // transpose and conj
        temp_reg[0] = (src_local_double_ptr[(1 * Nc + j) * 2 + 0] \
                    + (src_local_double_ptr[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local_double_ptr[(1 * Nc + j) * 2 + 1] \
                    + (src_local_double_ptr[(3 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local_double_ptr[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local_double_ptr[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local_double_ptr[(j * Nc + i) * 2 + 0];
      }
      // dst_local[0 * Nc + i] += temp1;
      // dst_local[2 * Nc + i] += temp1 * flag;
      dst_local_double_ptr[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local_double_ptr[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 0] += flag * temp_res1[0];
      dst_local_double_ptr[(2 * Nc + i) * 2 + 1] += flag * temp_res1[1];
  
      // dst_local[1 * Nc + i] += temp2;
      // dst_local[3 * Nc + i] += temp2 * flag;
      dst_local_double_ptr[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local_double_ptr[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 0] += flag * temp_res2[0];
      dst_local_double_ptr[(3 * Nc + i) * 2 + 1] += flag * temp_res2[1];
    }
#endif
  }

  Complex* dst_global = p.getPointVector(static_cast<Complex *>(fermion_out), Lx, Ly, Lz, Lt);
  for (int i = 0; i < Ns * Nc; i++) {
    dst_global[i] = dst_local[i];
  }
  
}

void NewDslash::calculateDslash(int invert_flag) {
  int Lx = dslashParam_->Lx;
  int Ly = dslashParam_->Ly;
  int Lz = dslashParam_->Lz;
  int Lt = dslashParam_->Lt;
  int parity = dslashParam_->parity;
  double flag;
  if (invert_flag == 0) {
    flag = 1.0;
  } else {
    flag = -1.0;
  }

  int space = Lx * Ly * Lz * Lt >> 1;
  dim3 gridDim(space / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);

  checkCudaErrors(hipDeviceSynchronize());

  mpi_comm->preDslash(dslashParam_->fermion_in, parity, invert_flag);

  auto start = std::chrono::high_resolution_clock::now();
  void *args[] = {&dslashParam_->gauge, &dslashParam_->fermion_in, &dslashParam_->fermion_out, &Lx, &Ly, &Lz, &Lt, &parity, &grid_x, &grid_y, &grid_z, &grid_t, &flag};

  checkCudaErrors(hipLaunchKernel((void *)mpiDslashNew, gridDim, blockDim, args));
  checkCudaErrors(hipDeviceSynchronize());

  // boundary calculate
  mpi_comm->postDslash(dslashParam_->fermion_out, parity, invert_flag);
  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("naive total time: (without malloc free memcpy) : %.9lf sec, block size = %d\n", double(duration) / 1e9, BLOCK_SIZE);
}


void callNewDslash(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity, int invert_flag) {
  // int Lx = param->lattice_size[0];
  // int Ly = param->lattice_size[1];
  // int Lz = param->lattice_size[2];
  // int Lt = param->lattice_size[3];
  // int vol = Lx * Ly * Lz * Lt;
  DslashParam dslash_param(fermion_in, fermion_out, gauge, param, parity);
  NewDslash dslash_solver(dslash_param);
  dslash_solver.calculateDslash(0);
}
