#include "hip/hip_runtime.h"
// #include "qcu_wilson_dslash_neo.cuh"
#include "qcu_complex.cuh"
#include "qcu_point.cuh"
#include "qcu_communicator.cuh"
#include "qcu_shift_storage_complex.cuh"
#include <chrono>
#include "qcu_wilson_dslash_new_new.cuh"

#define INCLUDE_COMPUTATION




extern int grid_x;
extern int grid_y;
extern int grid_z;
extern int grid_t;
extern MPICommunicator *mpi_comm;
static bool memory_allocated_new;
extern void *qcu_gauge;
static void* coalesced_fermion_in;
static void* coalesced_fermion_out;
// static void* coalesced_gauge;
// extern void* qcu_gauge;




static __device__ __forceinline__ void reconstructSU3(double *su3)
{
  // su3[6] = (su3[1] * su3[5] - su3[2] * su3[4]).conj();
  // su3[7] = (su3[2] * su3[3] - su3[0] * su3[5]).conj();
  // su3[8] = (su3[0] * su3[4] - su3[1] * su3[3]).conj();
  su3[6 * 2 + 0] = (su3[1 * 2 + 0] * su3[5 * 2 + 0] - su3[1 * 2 + 1] * su3[5 * 2 + 1]) \
                 - (su3[2 * 2 + 0] * su3[4 * 2 + 0] - su3[2 * 2 + 1] * su3[4 * 2 + 1]);
  su3[6 * 2 + 1] = (su3[2 * 2 + 1] * su3[4 * 2 + 0] + su3[2 * 2 + 0] * su3[4 * 2 + 1]) \
                 - (su3[1 * 2 + 1] * su3[5 * 2 + 0] + su3[1 * 2 + 0] * su3[5 * 2 + 1]); // conj()

  su3[7 * 2 + 0] = (su3[2 * 2 + 0] * su3[3 * 2 + 0] - su3[2 * 2 + 1] * su3[3 * 2 + 1]) \
                 - (su3[0 * 2 + 0] * su3[5 * 2 + 0] - su3[0 * 2 + 1] * su3[5 * 2 + 1]);
  su3[7 * 2 + 1] = (su3[0 * 2 + 1] * su3[5 * 2 + 0] + su3[0 * 2 + 0] * su3[5 * 2 + 1]) \
                 - (su3[2 * 2 + 1] * su3[3 * 2 + 0] + su3[2 * 2 + 0] * su3[3 * 2 + 1]); // conj()

  su3[8 * 2 + 0] = (su3[0 * 2 + 0] * su3[4 * 2 + 0] - su3[0 * 2 + 1] * su3[4 * 2 + 1]) \
                 - (su3[1 * 2 + 0] * su3[3 * 2 + 0] - su3[1 * 2 + 1] * su3[3 * 2 + 1]);
  su3[8 * 2 + 1] = (su3[1 * 2 + 1] * su3[3 * 2 + 0] + su3[1 * 2 + 0] * su3[3 * 2 + 1]) \
                 - (su3[0 * 2 + 1] * su3[4 * 2 + 0] + su3[0 * 2 + 0] * su3[4 * 2 + 1]); // conj()
}



__device__ __forceinline__ void loadGauge(double* u_local, void* gauge_ptr, int direction, const Point& p, int Lx, int Ly, int Lz, int Lt) {
  double* u = reinterpret_cast<double*>(p.getPointGauge(static_cast<Complex*>(gauge_ptr), direction, Lx, Ly, Lz, Lt));
  for (int i = 0; i < (Nc - 1) * Nc * 2; i++) {
    u_local[i] = u[i];
  }
  // reconstructSU3(u_local);
  reconstructSU3(reinterpret_cast<double*>(u_local));
}





__device__ __forceinline__ void loadVector(double* src_local, void* fermion_in, const Point& p, int Lx, int Ly, int Lz, int Lt) {
  // Complex* src = p.getPointVector(static_cast<Complex *>(fermion_in), Lx, Ly, Lz, Lt);
  double* src_double = reinterpret_cast<double*>(p.getPointVector(static_cast<Complex *>(fermion_in), Lx, Ly, Lz, Lt));

  for (int i = 0; i < Ns * Nc * 2; i++) {
    src_local[i] = src_double[i];
  }
}



static __global__ void mpiDslashNew(void *gauge, void *fermion_in, void *fermion_out,int Lx, int Ly, int Lz, int Lt, int parity, int grid_x, int grid_y, int grid_z, int grid_t, double flag_param) {
  assert(parity == 0 || parity == 1);
  int half_Lx = Lx >>= 1;

  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int t = thread_id / (Lz * Ly * half_Lx);
  int z = thread_id % (Lz * Ly * half_Lx) / (Ly * half_Lx);
  int y = thread_id % (Ly * half_Lx) / half_Lx;
  int x = thread_id % half_Lx;

  int coord_boundary;
  double flag = flag_param;


  Point p(x, y, z, t, parity);
  Point move_point;
  double u_local[Nc * Nc * 2];   // for GPU
  double src_local[Ns * Nc * 2]; // for GPU
  double dst_local[Ns * Nc * 2]; // for GPU

  double temp_reg[2];
  double temp_res1[2];
  double temp_res2[2];
  int eo = (y+z+t) & 0x01;

  for (int i = 0; i < Ns * Nc * 2; i++) {
    // dst_local[i].clear2Zero();
    dst_local[i] = 0;
  }

  // \mu = 1
  loadGauge(u_local, gauge, X_DIRECTION, p, half_Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, X_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadVector(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);
  // x front    x == half_Lx-1 && parity != eo
  coord_boundary = (grid_x > 1 && x == half_Lx-1 && parity != eo) ? half_Lx-1 : half_Lx;
  if (x < coord_boundary) {

    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    - (-src_local[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    - (src_local[(3 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];

        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    - (-src_local[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    - (src_local[(2 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];
      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 0] += -flag * temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 1] += flag * temp_res1[0];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 0] += -flag * temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 1] += flag * temp_res2[0];
    }

  }
  // x back   x==0 && parity == eo
  move_point = p.move(BACK, X_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadGauge(u_local, gauge, X_DIRECTION, move_point, half_Lx, Ly, Lz, Lt);
  loadVector(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);;

  coord_boundary = (grid_x > 1 && x==0 && parity == eo) ? 1 : 0;
  if (x >= coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    + (-src_local[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    + (src_local[(3 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];


        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    + (-src_local[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    + (src_local[(2 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];
      }
      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 0] += flag * temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 1] += -flag * temp_res1[0];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 0] += flag * temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 1] += -flag * temp_res2[0];
    }
  }

  // \mu = 2
  // y front
  loadGauge(u_local, gauge, Y_DIRECTION, p, half_Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, Y_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadVector(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_y > 1) ? Ly-1 : Ly;
  if (y < coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    + (src_local[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    + (src_local[(3 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];


        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    - (src_local[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    - (src_local[(2 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];

      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 0] += flag * temp_res1[0];
      dst_local[(3 * Nc + i) * 2 + 1] += flag * temp_res1[1];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 0] += -flag * temp_res2[0];
      dst_local[(2 * Nc + i) * 2 + 1] += -flag * temp_res2[1];
    }
  }

  // y back
  move_point = p.move(BACK, Y_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadGauge(u_local, gauge, Y_DIRECTION, move_point, half_Lx, Ly, Lz, Lt);
  loadVector(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_y > 1) ? 1 : 0;
  if (y >= coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    - (src_local[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    - (src_local[(3 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];

        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    + (src_local[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    + (src_local[(2 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];

      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 0] += -flag * temp_res1[0];
      dst_local[(3 * Nc + i) * 2 + 1] += -flag * temp_res1[1];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 0] += flag * temp_res2[0];
      dst_local[(2 * Nc + i) * 2 + 1] += flag * temp_res2[1];
    }
  }

  // \mu = 3
  // z front
  loadGauge(u_local, gauge, Z_DIRECTION, p, half_Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, Z_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadVector(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);
  coord_boundary = (grid_z > 1) ? Lz-1 : Lz;
  if (z < coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    - (-src_local[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    - (src_local[(2 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];

        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    + (-src_local[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    + (src_local[(3 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];
      }
      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 0] += -flag * temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 1] += flag * temp_res1[0];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 0] += flag * temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 1] += -flag * temp_res2[0];
    }
  }

  // z back
  move_point = p.move(BACK, Z_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadGauge(u_local, gauge, Z_DIRECTION, move_point, half_Lx, Ly, Lz, Lt);
  loadVector(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_z > 1) ? 1 : 0;
  if (z >= coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    + (-src_local[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    + (src_local[(2 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];


        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    - (-src_local[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    - (src_local[(3 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];
      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 0] += flag * temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 1] += -flag * temp_res1[0];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 0] += -flag * temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 1] += flag * temp_res2[0];
    }
  }

  // t: front
  loadGauge(u_local, gauge, T_DIRECTION, p, half_Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, T_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadVector(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_t > 1) ? Lt-1 : Lt;
  if (t < coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    - (src_local[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    - (src_local[(2 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];
        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    - (src_local[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    - (src_local[(3 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];
      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 0] += -flag * temp_res1[0];
      dst_local[(2 * Nc + i) * 2 + 1] += -flag * temp_res1[1];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 0] += -flag * temp_res2[0];
      dst_local[(3 * Nc + i) * 2 + 1] += -flag * temp_res2[1];
    }
  }
  // t: back
  move_point = p.move(BACK, T_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadGauge(reinterpret_cast<double*>(u_local), gauge, T_DIRECTION, move_point, half_Lx, Ly, Lz, Lt);
  loadVector(reinterpret_cast<double*>(src_local), fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_t > 1) ? 1 : 0;
  if (t >= coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    + (src_local[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    + (src_local[(2 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];

        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    + (src_local[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    + (src_local[(3 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];
      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 0] += flag * temp_res1[0];
      dst_local[(2 * Nc + i) * 2 + 1] += flag * temp_res1[1];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 0] += flag * temp_res2[0];
      dst_local[(3 * Nc + i) * 2 + 1] += flag * temp_res2[1];
    }
  }

  double* dst_global = reinterpret_cast<double*>(p.getPointVector(static_cast<Complex *>(fermion_out), half_Lx, Ly, Lz, Lt));
  for (int i = 0; i < Ns * Nc * 2; i++) {
    dst_global[i] = dst_local[i];
  }
}


static __device__ __forceinline__ void loadGaugeCoalesced(double* u_local, void* gauge_ptr, int direction, const Point& p, int sub_Lx, int Ly, int Lz, int Lt) {
  double* start_ptr = reinterpret_cast<double*>(p.getCoalescedGaugeAddr (gauge_ptr, direction, sub_Lx, Ly, Lz, Lt));
  int sub_vol = sub_Lx * Ly * Lz * Lt;
  for (int i = 0; i < (Nc - 1) * Nc; i++) {
    u_local[2*i] = start_ptr[0];
    u_local[2*i+1] = start_ptr[1];
    start_ptr += sub_vol * 2;
  }
  reconstructSU3(u_local);
}

static __device__ __forceinline__ void loadVectorCoalesced(double* src_local, void* fermion_in, const Point& p, int half_Lx, int Ly, int Lz, int Lt) {
  // Complex* start_ptr = p.getCoalescedVectorAddr (fermion_in, half_Lx, Ly, Lz, Lt);
  double* start_ptr = reinterpret_cast<double*>(p.getCoalescedVectorAddr (fermion_in, half_Lx, Ly, Lz, Lt));

  int sub_vol = half_Lx * Ly * Lz * Lt;

  for (int i = 0; i < Ns * Nc; i++) {
    src_local[2*i] = start_ptr[0];
    src_local[2*i+1] = start_ptr[1];
    start_ptr += sub_vol * 2;
  }
}

static __device__ __forceinline__ void storeVectorCoalesced(double* dst_local, void* fermion_out, const Point& p, int half_Lx, int Ly, int Lz, int Lt) {
  // Complex* start_ptr = p.getCoalescedVectorAddr (fermion_out, half_Lx, Ly, Lz, Lt);
  double* start_ptr = reinterpret_cast<double*>(p.getCoalescedVectorAddr (fermion_out, half_Lx, Ly, Lz, Lt));

  int sub_vol = half_Lx * Ly * Lz * Lt;

  for (int i = 0; i < Ns * Nc; i++) {
    start_ptr[0] = dst_local[2*i];
    start_ptr[1] = dst_local[2*i+1];
    start_ptr += sub_vol * 2;
  }
}

static __global__ void mpiDslashNewCoalesce(void *gauge, void *fermion_in, void *fermion_out,int Lx, int Ly, int Lz, int Lt, int parity, int grid_x, int grid_y, int grid_z, int grid_t, double flag_param) {
  assert(parity == 0 || parity == 1);
  int half_Lx = Lx >>= 1;

  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int t = thread_id / (Lz * Ly * half_Lx);
  int z = thread_id % (Lz * Ly * half_Lx) / (Ly * half_Lx);
  int y = thread_id % (Ly * half_Lx) / half_Lx;
  int x = thread_id % half_Lx;

  int coord_boundary;
  double flag = flag_param;


  Point p(x, y, z, t, parity);
  Point move_point;
  double u_local[Nc * Nc * 2];   // for GPU
  double src_local[Ns * Nc * 2]; // for GPU
  double dst_local[Ns * Nc * 2]; // for GPU

  double temp_reg[2];
  double temp_res1[2];
  double temp_res2[2];
  int eo = (y+z+t) & 0x01;

  for (int i = 0; i < Ns * Nc * 2; i++) {
    // dst_local[i].clear2Zero();
    dst_local[i] = 0;
  }

  // \mu = 1
  loadGaugeCoalesced(u_local, gauge, X_DIRECTION, p, half_Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, X_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadVectorCoalesced(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);
  // x front    x == half_Lx-1 && parity != eo
  coord_boundary = (grid_x > 1 && x == half_Lx-1 && parity != eo) ? half_Lx-1 : half_Lx;
  if (x < coord_boundary) {

    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    - (-src_local[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    - (src_local[(3 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];

        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    - (-src_local[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    - (src_local[(2 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];
      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 0] += -flag * temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 1] += flag * temp_res1[0];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 0] += -flag * temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 1] += flag * temp_res2[0];
    }

  }
  // x back   x==0 && parity == eo
  move_point = p.move(BACK, X_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadGaugeCoalesced(u_local, gauge, X_DIRECTION, move_point, half_Lx, Ly, Lz, Lt);
  loadVectorCoalesced(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);;

  coord_boundary = (grid_x > 1 && x==0 && parity == eo) ? 1 : 0;
  if (x >= coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    + (-src_local[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    + (src_local[(3 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];


        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    + (-src_local[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    + (src_local[(2 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];
      }
      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 0] += flag * temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 1] += -flag * temp_res1[0];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 0] += flag * temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 1] += -flag * temp_res2[0];
    }
  }

  // \mu = 2
  // y front
  loadGaugeCoalesced(u_local, gauge, Y_DIRECTION, p, half_Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, Y_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadVectorCoalesced(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_y > 1) ? Ly-1 : Ly;
  if (y < coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    + (src_local[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    + (src_local[(3 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];


        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    - (src_local[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    - (src_local[(2 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];

      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 0] += flag * temp_res1[0];
      dst_local[(3 * Nc + i) * 2 + 1] += flag * temp_res1[1];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 0] += -flag * temp_res2[0];
      dst_local[(2 * Nc + i) * 2 + 1] += -flag * temp_res2[1];
    }
  }

  // y back
  move_point = p.move(BACK, Y_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadGaugeCoalesced(u_local, gauge, Y_DIRECTION, move_point, half_Lx, Ly, Lz, Lt);
  loadVectorCoalesced(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_y > 1) ? 1 : 0;
  if (y >= coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    - (src_local[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    - (src_local[(3 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];

        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    + (src_local[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    + (src_local[(2 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];

      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(3 * Nc + i) * 2 + 0] += -flag * temp_res1[0];
      dst_local[(3 * Nc + i) * 2 + 1] += -flag * temp_res1[1];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(2 * Nc + i) * 2 + 0] += flag * temp_res2[0];
      dst_local[(2 * Nc + i) * 2 + 1] += flag * temp_res2[1];
    }
  }

  // \mu = 3
  // z front
  loadGaugeCoalesced(u_local, gauge, Z_DIRECTION, p, half_Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, Z_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadVectorCoalesced(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);
  coord_boundary = (grid_z > 1) ? Lz-1 : Lz;
  if (z < coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    - (-src_local[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    - (src_local[(2 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];

        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    + (-src_local[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    + (src_local[(3 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];
      }
      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 0] += -flag * temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 1] += flag * temp_res1[0];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 0] += flag * temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 1] += -flag * temp_res2[0];
    }
  }

  // z back
  move_point = p.move(BACK, Z_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadGaugeCoalesced(u_local, gauge, Z_DIRECTION, move_point, half_Lx, Ly, Lz, Lt);
  loadVectorCoalesced(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_z > 1) ? 1 : 0;
  if (z >= coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    + (-src_local[(2 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    + (src_local[(2 * Nc + j) * 2 + 0] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];


        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    - (-src_local[(3 * Nc + j) * 2 + 1] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    - (src_local[(3 * Nc + j) * 2 + 0] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];
      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 0] += flag * temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 1] += -flag * temp_res1[0];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 0] += -flag * temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 1] += flag * temp_res2[0];
    }
  }

  // t: front
  loadGaugeCoalesced(u_local, gauge, T_DIRECTION, p, half_Lx, Ly, Lz, Lt);
  move_point = p.move(FRONT, T_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadVectorCoalesced(src_local, fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_t > 1) ? Lt-1 : Lt;
  if (t < coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    - (src_local[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    - (src_local[(2 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res1[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];
        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    - (src_local[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    - (src_local[(3 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 0] \
                      - temp_reg[1] * u_local[(i * Nc + j) * 2 + 1];
        temp_res2[1] += temp_reg[0] * u_local[(i * Nc + j) * 2 + 1] \
                      + temp_reg[1] * u_local[(i * Nc + j) * 2 + 0];
      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 0] += -flag * temp_res1[0];
      dst_local[(2 * Nc + i) * 2 + 1] += -flag * temp_res1[1];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 0] += -flag * temp_res2[0];
      dst_local[(3 * Nc + i) * 2 + 1] += -flag * temp_res2[1];
    }
  }
  // t: back
  move_point = p.move(BACK, T_DIRECTION, half_Lx, Ly, Lz, Lt);
  loadGaugeCoalesced(reinterpret_cast<double*>(u_local), gauge, T_DIRECTION, move_point, half_Lx, Ly, Lz, Lt);
  loadVectorCoalesced(reinterpret_cast<double*>(src_local), fermion_in, move_point, half_Lx, Ly, Lz, Lt);

  coord_boundary = (grid_t > 1) ? 1 : 0;
  if (t >= coord_boundary) {
    #pragma unroll
    for (int i = 0; i < Nc; i++) {
      temp_res1[0] = temp_res1[1] = 0;
      temp_res2[0] = temp_res2[1] = 0;
      #pragma unroll
      for (int j = 0; j < Nc; j++) {
        // first row vector with col vector
        temp_reg[0] = (src_local[(0 * Nc + j) * 2 + 0] \
                    + (src_local[(2 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(0 * Nc + j) * 2 + 1] \
                    + (src_local[(2 * Nc + j) * 2 + 1] * flag));

        temp_res1[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res1[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];

        // second row vector with col vector
        temp_reg[0] = (src_local[(1 * Nc + j) * 2 + 0] \
                    + (src_local[(3 * Nc + j) * 2 + 0] * flag));
        temp_reg[1] = (src_local[(1 * Nc + j) * 2 + 1] \
                    + (src_local[(3 * Nc + j) * 2 + 1] * flag));

        temp_res2[0] += temp_reg[0] * u_local[(j * Nc + i) * 2 + 0] \
                      - temp_reg[1] * (-u_local[(j * Nc + i) * 2 + 1]);
        temp_res2[1] += temp_reg[0] * (-u_local[(j * Nc + i) * 2 + 1]) \
                      + temp_reg[1] * u_local[(j * Nc + i) * 2 + 0];
      }

      dst_local[(0 * Nc + i) * 2 + 0] += temp_res1[0];
      dst_local[(0 * Nc + i) * 2 + 1] += temp_res1[1];
      dst_local[(2 * Nc + i) * 2 + 0] += flag * temp_res1[0];
      dst_local[(2 * Nc + i) * 2 + 1] += flag * temp_res1[1];

      dst_local[(1 * Nc + i) * 2 + 0] += temp_res2[0];
      dst_local[(1 * Nc + i) * 2 + 1] += temp_res2[1];
      dst_local[(3 * Nc + i) * 2 + 0] += flag * temp_res2[0];
      dst_local[(3 * Nc + i) * 2 + 1] += flag * temp_res2[1];
    }
  }

  // double* dst_global = reinterpret_cast<double*>(p.getPointVector(static_cast<Complex *>(fermion_out), half_Lx, Ly, Lz, Lt));
  // for (int i = 0; i < Ns * Nc * 2; i++) {
  //   dst_global[i] = dst_local[i];
  // }
  storeVectorCoalesced(dst_local, fermion_out, p, half_Lx, Ly, Lz, Lt);  // store result
}


void NewDslash::calculateDslashCoalesced(int invert_flag) {
  int Lx = dslashParam_->Lx;
  int Ly = dslashParam_->Ly;
  int Lz = dslashParam_->Lz;
  int Lt = dslashParam_->Lt;
  int parity = dslashParam_->parity;
  double flag;
  if (invert_flag == 0) {
    flag = 1.0;
  } else {
    flag = -1.0;
  }

  int space = Lx * Ly * Lz * Lt >> 1;
  dim3 gridDim(space / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);

  checkCudaErrors(hipDeviceSynchronize());

  mpi_comm->preDslash(dslashParam_->fermion_in, parity, invert_flag);

  auto start = std::chrono::high_resolution_clock::now();
  void *args[] = {&dslashParam_->gauge, &dslashParam_->fermion_in, &dslashParam_->fermion_out, &Lx, &Ly, &Lz, &Lt, &parity, &grid_x, &grid_y, &grid_z, &grid_t, &flag};

  checkCudaErrors(hipLaunchKernel((void *)mpiDslashNewCoalesce, gridDim, blockDim, args));
  checkCudaErrors(hipDeviceSynchronize());

  // boundary calculate
  mpi_comm->postDslash(dslashParam_->fermion_out, parity, invert_flag);
  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("coalescing without class total time: (without malloc free memcpy) : %.9lf sec, block size = %d\n", double(duration) / 1e9, BLOCK_SIZE);
}


void callNewDslashCoalesced(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity, int invert_flag) {
  int Lx = param->lattice_size[0];
  int Ly = param->lattice_size[1];
  int Lz = param->lattice_size[2];
  int Lt = param->lattice_size[3];
  int vol = Lx * Ly * Lz * Lt;

  if (!memory_allocated_new) {
    checkCudaErrors(hipMalloc(&coalesced_fermion_in, sizeof(double) * vol / 2 * Ns * Nc * 2));
    checkCudaErrors(hipMalloc(&coalesced_fermion_out, sizeof(double) * vol / 2 * Ns * Nc * 2));
    memory_allocated_new = true;
  }

  shiftVectorStorageTwoDouble(coalesced_fermion_in, fermion_in, TO_COALESCE, Lx, Ly, Lz, Lt);

  DslashParam dslash_param(coalesced_fermion_in, coalesced_fermion_out, qcu_gauge, param, parity);
  NewDslash dslash_solver(dslash_param);
  dslash_solver.calculateDslashCoalesced(0);

  shiftVectorStorageTwoDouble(fermion_out, coalesced_fermion_out, TO_NON_COALESCE, Lx, Ly, Lz, Lt);
}




void NewDslash::calculateDslash(int invert_flag) {
  int Lx = dslashParam_->Lx;
  int Ly = dslashParam_->Ly;
  int Lz = dslashParam_->Lz;
  int Lt = dslashParam_->Lt;
  int parity = dslashParam_->parity;
  double flag;
  if (invert_flag == 0) {
    flag = 1.0;
  } else {
    flag = -1.0;
  }

  int space = Lx * Ly * Lz * Lt >> 1;
  dim3 gridDim(space / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);

  checkCudaErrors(hipDeviceSynchronize());

  mpi_comm->preDslash(dslashParam_->fermion_in, parity, invert_flag);

  auto start = std::chrono::high_resolution_clock::now();
  void *args[] = {&dslashParam_->gauge, &dslashParam_->fermion_in, &dslashParam_->fermion_out, &Lx, &Ly, &Lz, &Lt, &parity, &grid_x, &grid_y, &grid_z, &grid_t, &flag};

  checkCudaErrors(hipLaunchKernel((void *)mpiDslashNew, gridDim, blockDim, args));
  checkCudaErrors(hipDeviceSynchronize());

  // boundary calculate
  mpi_comm->postDslash(dslashParam_->fermion_out, parity, invert_flag);
  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("naive without class total time: (without malloc free memcpy) : %.9lf sec, block size = %d\n", double(duration) / 1e9, BLOCK_SIZE);
}




void callNewDslash(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity, int invert_flag) {
  DslashParam dslash_param(fermion_in, fermion_out, gauge, param, parity);
  NewDslash dslash_solver(dslash_param);
  dslash_solver.calculateDslash(0);
}
